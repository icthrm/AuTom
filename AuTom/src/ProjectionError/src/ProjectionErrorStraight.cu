#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include "atom.h"
#include "mrc_file.h"
#define PI_180 0.01745329252f

using namespace std;
#ifndef PI
#define     PI  3.14159265358979323846
#endif

#define LOG_BLOCKDIM 6 //2^8=256
#define BLOCKDIM 64

float ANG[ANG_MAX];
float BNG[ANG_MAX];

__constant__ float p_ang[ANG_MAX];
__constant__ int c_Zmin[4096];
/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__device__ float lineindex_value(const float &ang, const int &x, const int &z,
		const int &z0, const int &x0) {
	return ((x - x0 + 0.5f) * __cosf(ang) + (z - z0 + 0.5f) * __sinf(ang) + x0);
//return ((x-x0+0.5f)*__cosf(ang)+(z-z0+0.5f)*__sinf(ang));
}
__device__ inline void atomicAddF(float* address, float value) {
	while (value)
		value = atomicExch(address, value + atomicExch(address, 0.0f));

}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
//SIRT
/**********************************************************************************/
__global__ void Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int AngN) //A*X
		{

	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	float ang, rf, weight;
	int r;
	m = n + slid.min;
	int pixel_allnum = slc.X * slc.Z;

	if (m < pixel_allnum) {
		z = (int) __fdividef(m, slc.X);
		x = m - z * slc.X;

		for (j = 0; j < AngN; j++) {
			ang = PI_180 * p_ang[j];
			rf = lineindex_value(ang, x, z, slc.z0, slc.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//r += slc.x0;

			if (r >= 0 && r < slc.X - 1) {
				row_index = j * slc.X + r;
				atomicAddF(&d_prj_num[row_index], weight);
				atomicAddF(&d_prj_num[row_index + 1], 1 - weight);
				atomicAddF(&d_prj_calc[row_index], weight * d_slc_data[n]);
				atomicAddF(&d_prj_calc[row_index + 1],
						(1 - weight) * d_slc_data[n]);
			} //end if weight
			else if (r == slc.X - 1) //left
					{
				row_index = j * slc.X + r;
				atomicAddF(&d_prj_num[row_index], weight);
				atomicAddF(&d_prj_calc[row_index], weight * d_slc_data[n]);
			} else if (r == -1) //right
					{
				row_index = j * slc.X;
				atomicAddF(&d_prj_num[row_index], 1 - weight);
				atomicAddF(&d_prj_calc[row_index],
						(1 - weight) * d_slc_data[n]);
			}
		} //end for j
	} //end if m
}
/**********************************************************************************/
__global__ void proj_calc_sub(float* d_prj_real, float *d_prj_calc,
		float *d_prj_num, int line_allnum) {
	const unsigned int r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	float s, c;
	s = d_prj_calc[r];
	c = d_prj_num[r];
	if ((r < line_allnum) && (c != 0)) {
		atomicAddF(&d_prj_calc[r], (-1) * d_prj_real[r]);
	}
}
__global__ void proj_calc_div(float *d_prj_calc, float *d_prj_num,
		int line_allnum) {
	const unsigned int r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	float s, c;
	s = d_prj_calc[r];
	c = d_prj_num[r];
	if ((r < line_allnum) && (c != 0)) {
		d_prj_calc[r] = __fdividef(s, c);
	}
}
/**********************************************************************************/
__global__ void SIRT_one_update(float *d_prj_real, float *d_prj_calc,
		float *d_slc_data, int AngN, Slice slc, Slide slid,
		float SIRT_ITR_STEP) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	float c = 0.0f;
	float s = 0.0f;
	float ang, rf, weight;
	int r;
	m = n + slid.min;
	int pixel_allnum = slc.X * slc.Z;

	if (m < pixel_allnum) {
		z = (int) __fdividef(m, slc.X);
		x = m - z * slc.X;
		for (j = 0; j < AngN; j++) {
			ang = PI_180 * p_ang[j];
			rf = lineindex_value(ang, x, z, slc.z0, slc.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			r += slc.x0;
			if (r >= 0 && r < slc.X - 1) {
				c++;
				row_index = j * slc.X + r;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
				s +=
						(1 - weight)
								* (d_prj_real[row_index + 1]
										- d_prj_calc[row_index + 1]);
			} //end if
			else if (r == slc.X - 1) {
				c += weight;
				row_index = j * slc.X + r;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
			} else if (r == -1) {
				c += 1 - weight;
				row_index = j * slc.X;
				s += (1 - weight)
						* (d_prj_real[row_index] - d_prj_calc[row_index]);
			}
		} //end for j

		if (c != 0)
			d_slc_data[n] += __fdividef(s, c) * SIRT_ITR_STEP;

	} //end if m
}
/**********************************************************************************/
//update the pixel value of slc by comparing the value of real_gray and calc_gray of prj;
__host__ int SIRT_update_slice(float *d_prj_real, float *d_prj_calc,
		float *d_prj_num, float *d_slc_data, int AngN, Slice slc, Slide slid,
		int pixel_num, float SIRT_ITR_STEP)

		{
	int line_allnum = slc.X * AngN;
	int num_blocks;

	unsigned int timer = 0;

	num_blocks = pixel_num >> LOG_BLOCKDIM;

	Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,
			d_slc_data, slc, slid, AngN);

	num_blocks = line_allnum >> LOG_BLOCKDIM;
	proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,
			line_allnum);

	hipDeviceSynchronize();
	num_blocks = pixel_num >> LOG_BLOCKDIM;

	SIRT_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_real,
			d_prj_calc, d_slc_data, AngN, slc, slid, SIRT_ITR_STEP);
	hipDeviceSynchronize();
	//CUT_SAFE_CALL( cutStopTimer( timer));
//   printf("Processing time: %f (ms) in Slice_one_update\n", cutGetTimerValue( timer));

	return TRUE;
}

__global__ void Slice_backproj(float *d_prj_real, float *d_slc_data, int AngN,
		Slice slc, Slide slid) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	float c = 0.0f;
	float s = 0.0f;
	float ang, rf, weight;
	int r;
	m = n + slid.min;
	int pixel_allnum = slc.X * slc.Z;

	if (m < pixel_allnum) {
		z = (int) __fdividef(m, slc.X);
		x = m - z * slc.X;
		for (j = 0; j < AngN; j++) {
			ang = PI_180 * p_ang[j];
			rf = lineindex_value(ang, x, z, slc.z0, slc.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			if (r >= 0 && r < slc.X - 1) {
				c++;
				row_index = j * slc.X + r;
				s += weight * d_prj_real[row_index]
						+ (1 - weight) * d_prj_real[row_index + 1];
			} else if (r == slc.X - 1) {
				c += weight;
				row_index = j * slc.X + r;
				s += weight * d_prj_real[row_index];
			} else if (r == -1) {
				c += (1 - weight);
				row_index = j * slc.X;
				s += (1 - weight) * d_prj_real[row_index];
			}

		} //end for j

		if (c != 0)
			d_slc_data[n] = __fdividef(s, c);

	} //end if m

}
void help() {
	printf("\n usage:\n");
	printf(" atom <inputfile> <outputfile> [options]\n");
	printf(
			"     art/ART or sirt/SIRT must be entered so the program can decide wich method to be used \n");
	printf(
			"     [-n]: the interative number,default is 10\n           if n is 0,this program will just do a simple backprojection\n");
	printf(
			"     [-t]: the step size for refinement, suggested value is about 0.3,which is also the default\n");
	printf(
			"     example: atom caveolae.mrc caveolae_rec.mrc SIRT -n 50 -t 0.2 \n\n");
}
int read_proj(FILE *fin, MrcHeader *head, int slcN, float *prj_real) {
	float *slcdata;
	int dataN, n;
	dataN = head->nx * head->nz;

	if ((slcdata = (float *) malloc(sizeof(float) * dataN)) == NULL) {
		printf(
				"Error with function 'read_prj()'! Can not malloc memeroy for slcdata!");
		return FALSE;
	}

	mrc_read_slice(fin, head, slcN, 'y', slcdata);
	for (n = 0; n < dataN; n++)
		prj_real[n] = slcdata[n];

	free(slcdata);

	return TRUE;
}
void totalCount(char* filename, float*a, int num) {
	FILE *result_file;
/*	
	string ss=filename;
	ss="result_"+ss+".txt";*/
	
	if ((result_file = fopen("result.txt", "at+")) == NULL) {
		printf("Cannot open file strike any key exit!\n");
	}
	//fprintf(result_file,
	//			"filename	mean	absolute mean	standard deviation	\n\n");
	double meanf = 0.0;
	double absf = 0.0;
	double stdf = 0.0;
	double subf = 0.0;
	for (int i = 0; i < num; i++) {
		subf = a[i];
		meanf += subf;
		absf += fabs(subf);
		stdf += subf * subf;
	}
	meanf = meanf / num;
	absf = absf / num;
	stdf = sqrt(stdf) / num;
	fprintf(result_file,
			"filename %s, mean %f,absolute mean %f ,standard deviation %f\n\n",
			filename, meanf, absf, stdf);
}
void totalCount(char* filename, float*a, float *b, int num) {
	FILE *result_file;
	if ((result_file = fopen("result.txt", "at+")) == NULL) {
		printf("Cannot open file strike any key exit!\n");
	}
	double meanf = 0.0;
	double absf = 0.0;
	double stdf = 0.0;
	double subf = 0.0;
	for (int i = 0; i < num; i++) {
		subf = a[i] - b[i];
		meanf += subf;
		absf += fabs(subf);
		stdf += subf * subf;
	}
	meanf = meanf / num;
	absf = absf / num;
	stdf = sqrt(stdf) / num;
	fprintf(result_file,
			"filename %s, mean %f,absolute mean %f ,standard deviation %f\n\n",
			filename, meanf, absf, stdf);
}
/**********************************************************************************/
//int  ATOM(char *inf,char *outf, float ATOM_ITR_STEP, int ATOM_ITR_NUM,char *Method) //Method ="SIRT" means sirt, Method = "ART" means ART;Method ="SART" means sart
//count the projection error
/**'/home/ict/Public/ATOM-test/20130608experiment/mitochodria_ASART102.mrc' mitochondria_fin.mrc mitochondria_new.tlt  -t 286 -z -17.6 -f 0.87  -p 3.16
 */
int main(int argc, char *argv[]) {
//example: BBa_ASART102.mrc BBa_fin.mrc BBa_err.mrc BBa_rePro.mrc BBa_new.tlt  -t 256 -z 0.0 -f 0.29 -p 1.16
//input: 3D reconstruction result
//input: 2D original projection
//input: angle file
//input: offset e.g. -f 0.032
//output : reprojection mrc file  and projection error mrc file
//output : the mean error | the absolute mean error| the standard deviation
	char *prof;
	prof = (char *) malloc(TEXT_LINE_MAX);
	char *reconf;
	reconf = (char *) malloc(TEXT_LINE_MAX);
	char *angf;
	angf = (char *) malloc(TEXT_LINE_MAX);
// output mrc
	char * newprof;
	newprof = (char *) malloc(TEXT_LINE_MAX);
	char * errf;
	errf = (char *) malloc(TEXT_LINE_MAX);

	float offset, zshift, pitch;
	int thickness;
	reconf = argv[1];
	prof = argv[2];
	angf = argv[3];
	newprof = argv[4];
	errf = argv[5];

	for (int i = 6; i < argc; i = i + 2) {
		if (argv[i][0] != '-') {
			printf("\"%s\" error! No such option!\n", argv[i]);
			return 0;
		}

		switch (argv[i][1]) {
		case 'f':
			offset = atof(argv[i + 1]);
			break;
		case 't':
			thickness = atoi(argv[i + 1]);
			break;
		case 'z':
			zshift = atof(argv[i + 1]);
			break;
		case 'p':
			pitch = atof(argv[i + 1]);
			break;
		}
	}
	printf("start: %s %s %s: f:%f t:%d z:%f p:%f\n", reconf, prof, angf, offset,
			thickness, zshift, pitch);
// read in the projection &reconstruction result
	// open file
	FILE *pro_file, *recon_file, *ang_file, *newpro_file, *err_file;

	if ((pro_file = fopen(prof, "r")) == NULL) {
		printf("pro_file:Cannot open file strike any key exit!\n");
		return FALSE;
	}
	if ((recon_file = fopen(reconf, "r")) == NULL) {
		printf("recon_file:Cannot open file strike any key exit!\n");
		return FALSE;
	}
	if ((ang_file = fopen(angf, "r")) == NULL) {
		printf("ang_file:Cannot open file strike any key exit!\n");
		return FALSE;
	}
	if ((newpro_file = fopen(newprof, "w+")) == NULL) {
		printf("newpro_file:Cannot open file strike any key exit!\n");
	}
	if ((err_file = fopen(errf, "w+")) == NULL) {
		printf("err_file:Cannot open file strike any key exit!\n");
	}
	//get file information

	MrcHeader *pro_head, *recon_head, *newpro_head, *err_head;
	pro_head = (MrcHeader *) malloc(sizeof(MrcHeader));
	newpro_head = (MrcHeader *) malloc(sizeof(MrcHeader));
	err_head = (MrcHeader *) malloc(sizeof(MrcHeader));
	recon_head = (MrcHeader *) malloc(sizeof(MrcHeader));
	mrc_init_head(newpro_head);
	mrc_init_head(err_head);
	mrc_read_head(pro_file, pro_head);
	mrc_read_head(recon_file, recon_head);
	int pro_num = pro_head->nx * pro_head->ny * pro_head->nz;
	err_head->mode = newpro_head->mode = MRC_MODE_FLOAT;
	err_head->nx = newpro_head->nx = pro_head->nx;
	err_head->ny = newpro_head->ny = pro_head->ny;
	err_head->nz = newpro_head->nz = pro_head->nz;
	mrc_write_head(newpro_file, newpro_head);
	mrc_write_head(err_file, err_head);

	// need to release after debug
	float *newpro_data, *pro_data;
	//locate the new projection
	if ((newpro_data = (float *) malloc(sizeof(float) * pro_num)) == NULL) {
		printf(
				"Error with Function 'ATOM()'!Can't malloc memery for 'prj_real'!");
		return 0;
	}
	if ((pro_data = (float *) malloc(sizeof(float) * pro_num)) == NULL) {
		printf(
				"Error with Function 'ATOM()'!Can't malloc memery for 'prj_real'!");
		return 0;
	}
	mrc_read_all(pro_file, pro_head, pro_data);

	//handle the ang file
	int ang_num = 0;
	char* lstr, *tmp;
	lstr = (char *) malloc(TEXT_LINE_MAX);

	float *thita;
	thita = (float *) malloc(ANG_MAX * sizeof(float));

	while (NULL != (fgets(lstr, TEXT_LINE_MAX, ang_file))) {
		tmp = strtok(lstr, "\n");
		thita[ang_num++] = atof(tmp) + offset;
	}
	for (int i = 0; i < ang_num; i++) {
		ANG[i] = thita[i];
	}

	//for (int i = 0; i < ang_num; i++) {
	//	BNG[i] = -ANG[i];
	//}
	//count the coefficient
	Volume vol;
	vol.X = pro_head->nx;
	vol.Y = pro_head->ny;
	vol.Z = thickness;
	vol.zshift = zshift;
	vol.offset = offset;
	vol.pitch = pitch;

	Slice slc;
	Slide slid;
	slc.X = vol.X;
	slc.x0 = vol.X / 2;
	if (vol.pitch >= 0)
		slc.Z = (int) (vol.Z + tan(rad(vol.pitch)) * vol.Y);
	else
		slc.Z = (int) (vol.Z + tan(rad(-vol.pitch)) * vol.Y);
	slc.z0 = slc.Z / 2 + vol.zshift;

	int Z_minS, Z_maxS, Z_minE, Z_maxE;
	if (vol.pitch >= 0) {
		Z_minS = 0;
		Z_maxS = vol.Z;
		Z_minE = (int) (tan(rad(vol.pitch)) * vol.Y);
		Z_maxE = Z_minE + vol.Z;
	}

	if (vol.pitch < 0) {
		Z_minS = (int) (tan(rad(-vol.pitch)) * vol.Y);
		Z_maxS = vol.Z + Z_minS;
		Z_minE = 0;
		Z_maxE = vol.Z;
	}

	int line_allnum = vol.X * ang_num;
	int pixel_num = slc.X * vol.Z; //write pixels in fact per slice
	int num_blocks = pixel_num >> LOG_BLOCKDIM;
	//printf("ang %d line_allnum %d pixel_num %d num_blocks %d \n",ang_num,line_allnum,pixel_num,num_blocks);

	float *prj_real, *prj_calc, *slc_data;
	prj_real = (float *) malloc(sizeof(float) * line_allnum); //original projection
	memset(prj_real, 0, sizeof(float) * line_allnum);
	prj_calc = (float *) malloc(sizeof(float) * line_allnum); //counted projection
	memset(prj_calc, 0, sizeof(float) * line_allnum);
	slc_data = (float *) malloc(sizeof(float) * pixel_num); // the 3D slice for a y
	memset(slc_data, 0, sizeof(float) * pixel_num);

	float *d_prj_real;
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &d_prj_real, sizeof(float) * line_allnum));
	float *d_prj_calc;
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &d_prj_calc, sizeof(float) * line_allnum));
	CUDA_CHECK_RETURN(hipMemset(d_prj_calc, 0, sizeof(float) * line_allnum));
	float *d_prj_num;
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &d_prj_num, sizeof(float) * line_allnum));
	CUDA_CHECK_RETURN(hipMemset(d_prj_num, 0, sizeof(float) * line_allnum));
	float *d_slc_data;
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &d_slc_data, sizeof(float) * pixel_num));
	CUDA_CHECK_RETURN(hipMemset(d_slc_data, 0, sizeof(float) * pixel_num));

// test: output the 3Dresult
	/*FILE* re_file;
	 char * ref = "newrecon.mrc";
	 if ((re_file = fopen(ref, "w+")) == NULL) {
	 printf("Cannot open file strike any key exit!\n");
	 }
	 MrcHeader *re_head;
	 re_head = (MrcHeader *) malloc(sizeof(MrcHeader));
	 mrc_init_head(re_head);
	 re_head->mode = MRC_MODE_FLOAT;
	 re_head->nx = recon_head->nx;
	 re_head->ny = recon_head->ny;
	 re_head->nz = recon_head->nz;
	 mrc_write_head(re_file, re_head);
	 float * recon_data;
	 int total_num = vol.X * vol.Z * vol.Y;
	 recon_data = (float *) malloc(sizeof(float) * total_num); //original projection
	 */

	hipMemcpyToSymbol(HIP_SYMBOL(p_ang), ANG, sizeof(float) * ang_num);
	//hipMemcpyToSymbol(HIP_SYMBOL(p_ang), ANG, sizeof(float) * ang_num);//????? why we need to use this negitive angle
	// allocate device memory for result
	CUDA_CHECK_RETURN(hipGetLastError());
	for (int j = 0; j < vol.Y; j++) {
		//printf("j=%d\n", j);
		mrc_read_slice_y(pro_file, pro_head, j, prj_real);
		//mrc_write_slice(newpro_file, newpro_head, j, 'y', prj_real);
		hipMemset(d_prj_calc, 0, sizeof(float) * line_allnum);
		hipMemset(d_prj_num, 0, sizeof(float) * line_allnum);
		CUDA_CHECK_RETURN(
				hipMemcpy(d_prj_real, prj_real, sizeof(float) * line_allnum, hipMemcpyHostToDevice));

		mrc_read_slice_y(recon_file, recon_head, j, slc_data); //read the reconstruction data
		CUDA_CHECK_RETURN(
				hipMemcpy(d_slc_data, slc_data, sizeof(float) * pixel_num, hipMemcpyHostToDevice));
		slid.min = slc.X * ((j * (Z_minE - Z_minS)) / vol.Y + Z_minS);
		slid.max = slc.X * ((j * (Z_maxE - Z_maxS)) / vol.Y + Z_maxS);
		num_blocks = pixel_num >> LOG_BLOCKDIM;
		Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, d_slc_data, slc, slid, ang_num);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = line_allnum >> LOG_BLOCKDIM;
		proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, line_allnum);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(
				hipMemcpy((void*) prj_calc, (void*) d_prj_calc, sizeof(float) * line_allnum, hipMemcpyDeviceToHost));
		mrc_write_slice(newpro_file, newpro_head, j, 'y', prj_calc);
		proj_calc_sub<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_real,
				d_prj_calc, d_prj_num, line_allnum);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipGetLastError());
		//copy this part to the array
		CUDA_CHECK_RETURN(
				hipMemcpy((void*) prj_calc, (void*) d_prj_calc, sizeof(float) * line_allnum, hipMemcpyDeviceToHost));
		mrc_write_slice(err_file, err_head, j, 'y', prj_calc);
	}
	mrc_update_head(newprof);
	mrc_update_head(errf);
	//mrc_read_all(newpro_file, newpro_head, newpro_data);
	mrc_read_all(err_file, err_head, newpro_data);
	//totalCount(reconf, newpro_data, pro_data, pro_num);
	totalCount(reconf, newpro_data,pro_num);
	hipFree(d_prj_real);
	hipFree(d_prj_calc);
	hipFree(d_prj_num);
	hipFree(d_slc_data);

	free(slc_data);
	free(prj_calc);
	free(prj_real);
	free(pro_data);
	free(newpro_data);

	//fclose(re_file);
	fclose(pro_file);
	fclose(recon_file);
	fclose(ang_file);
	return 0;
}
