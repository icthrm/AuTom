#include "hip/hip_runtime.h"
/*
 * mrc_file.c
 *
 *  Created on: Apr 16, 2013
 *      Author: ict
 */
#include "mrc_file.h"

float rad(float ang) {
	return ang * PI / 180.0;
}

long get_file_size(FILE *fin) {
	fseek(fin, 0, SEEK_END);

	return ftell(fin);

}

/*******************************************************************************************/
int mrc_read_head(FILE *fin, MrcHeader *head) {
	if (ftello64(fin) != 0)
		rewind(fin);
	fread(head, 1024, 1, fin);

	/*if(!(head->cmap[0]=='M'&&head->cmap[1]=='A'&&head->cmap[2]=='P'))
	 {
	 printf("Error with function 'mrc_read_head()'! Warning: Not MRC format! \n");
	 return -1;
	 }*/

	return 0;
}

/*******************************************************************************************/
int mrc_write_head(FILE *fout, MrcHeader *head) {
	if (ftello64(fout) != 0)
		rewind(fout);
	if (!head
			|| !(head->cmap[0] == 'M' && head->cmap[1] == 'A'
					&& head->cmap[2] == 'P')) {
		printf(
				"Error with function 'mrc_write_head()'! Can not write the MrcHeader!");
		return -1;
	}
	fwrite(head, 1024, 1, fout);

	return 0;
}

/*******************************************************************************************/
int mrc_init_head(MrcHeader *head) {
	head->nx = 0;
	head->ny = 0;
	head->nz = 0;

	head->mode = MRC_MODE_FLOAT;

	head->nxstart = 0;
	head->nystart = 0;
	head->nzstart = 0;

	head->mx = 1;
	head->my = 1;
	head->mz = 1;

	head->xlen = 1;
	head->ylen = 1;
	head->zlen = 1;

	head->alpha = 90;
	head->beta = 90;
	head->gamma = 90;

	head->mapc = 1;
	head->mapr = 2;
	head->maps = 3;

	head->amin = 0;
	head->amax = 255;
	head->amean = 128;

	head->ispg = 0;
	head->nsymbt = 0;

	head->next = 0;

	head->creatid = 1000;
	head->cmap[0] = 'M';
	head->cmap[1] = 'A';
	head->cmap[2] = 'P';

	head->stamp[0] = 'D';
	return 0;
}

/*******************************************************************************************/
int mrc_replace_head(char *outf, MrcHeader *head) {
	FILE *fout;
	if ((fout = fopen(outf, "r+")) == NULL) {
		printf("Cannot open file strike any key exit!\n");
	}
	mrc_write_head(fout, head);
	fclose(fout);
	return 0;
}

/*******************************************************************************************/
int mrc_update_head(char *inoutf) {

	MrcHeader *head;
	head = (MrcHeader *) malloc(sizeof(MrcHeader));

	FILE *finout;

	if ((finout = fopen(inoutf, "r+")) == NULL) {
		printf("Cannot open file strike any key exit!\n");
	}

	mrc_read_head(finout, head);

	if (!head
			|| !(head->cmap[0] == 'M' && head->cmap[1] == 'A'
					&& head->cmap[2] == 'P')) {
		printf("Fatal erro! The file %s is not a valid mrc file!\n", inoutf);
		return -1;
	}

	float sum;
	float amin, amax, amean;
	int k, pNum;
	unsigned long site;
	unsigned char *p_uchar;
	short *p_short;
	float *p_float;

	fseek(finout, 1024, SEEK_SET);

	switch (head->mode) { //switch start

	/**********case MRC_MODE_BYTE ***********/
	case MRC_MODE_BYTE:

		pNum = head->nx * head->ny;

		if ((p_uchar = (unsigned char *) malloc(sizeof(unsigned char) * pNum))
				== NULL) {
			printf("Function 'malloc' erro, while updating head!\n");
			return -1;
		}

		printf("updating head!\n");
		fread(p_uchar, sizeof(unsigned char), pNum, finout);

		amin = amax = amean = p_uchar[0];
		sum = 0;
		for (site = 1; site < pNum; site++) {
			if (p_uchar[site] > amax)
				amax = p_uchar[site];
			if (p_uchar[site] < amin)
				amin = p_uchar[site];
			amean = amean + p_uchar[site];
		}
		amean /= pNum;
		sum = amean;
		for (k = 1; k < head->nz; k++) {
			amean = 0;
			fread(p_uchar, sizeof(unsigned char), pNum, finout);

			for (site = 0; site < pNum; site++) {
				if (p_uchar[site] > amax)
					amax = p_uchar[site];
				if (p_uchar[site] < amin)
					amin = p_uchar[site];
				amean = amean + p_uchar[site];
			}
			amean /= pNum;
			sum += amean;
		}
		amean = sum / head->nz;

		free(p_uchar);

		break;

		/**********case MRC_MODE_SHORT ***********/
	case MRC_MODE_SHORT:

		pNum = head->nx * head->ny;

		if ((p_short = (short *) malloc(sizeof(short) * pNum)) == NULL) {
			printf("Function 'malloc' erro, while updating head!\n");

			return -1;
		}

		printf("updating head!\n");

		fread(p_short, sizeof(short), pNum, finout);
		amin = amax = amean = p_short[0];
		sum = 0;
		for (site = 1; site < pNum; site++) {
			if (p_short[site] > amax)
				amax = p_short[site];
			if (p_short[site] < amin)
				amin = p_short[site];
			amean = amean + p_short[site];
		}
		amean /= pNum;
		sum = amean;
		for (k = 1; k < head->nz; k++) {
			amean = 0;
			fread(p_short, sizeof(short), pNum, finout);

			for (site = 0; site < pNum; site++) {
				if (p_short[site] > amax)
					amax = p_short[site];
				if (p_short[site] < amin)
					amin = p_short[site];
				amean = amean + p_short[site];
			}
			amean /= pNum;
			sum += amean;
		}
		amean = sum / head->nz;

		free(p_short);

		break;

		/**********case MRC_MODE_FLOAT ***********/
	case MRC_MODE_FLOAT:

		pNum = head->nx * head->ny;
		if ((p_float = (float *) malloc(sizeof(float) * pNum)) == NULL) {

			printf("Function malloc erro!\n");
			return -1;
		}

		printf("FLOAT:updating head!\n");

		fread(p_float, sizeof(float), pNum, finout);

		amin = amax = amean = p_float[0];
		sum = 0;
		for (site = 1; site < pNum; site++) {
			//printf("site :%d %d \n",pNum,site);
			if (p_float[site] > amax)
				amax = p_float[site];
			if (p_float[site] < amin)
				amin = p_float[site];
			amean = amean + p_float[site];
		}
		amean /= pNum;
		sum = amean;
		for (k = 1; k < head->nz; k++) {
			//printf("k :%d %d \n",head->nz,k);
			amean = 0;
			fread(p_float, sizeof(float), pNum, finout);

			for (site = 0; site < pNum; site++) {
				if (p_float[site] > amax)
					amax = p_float[site];
				if (p_float[site] < amin)
					amin = p_float[site];
				amean = amean + p_float[site];
			}
			amean /= pNum;
			sum += amean;
		}
		amean = sum / head->nz;

		free(p_float);

		break;
	} //switch end

	head->amin = amin;
	head->amax = amax;
	head->amean = amean;

	fclose(finout);

	mrc_replace_head(inoutf, head);
	free(head);
	printf("updating finished!\n");
	return 0;
}

/****************************************************************************************/
int mrc_new_file(char *newf, MrcHeader *outhead) {
	FILE *fnew;
	if ((fnew = fopen(newf, "w+")) == NULL) {
		printf("\nCannot open file strike any key exit!");
		return false;
	}
	mrc_write_head(fnew, outhead);
	fclose(fnew);
	return true;
}

/*******************************************************************************************/

int mrc_read_pixel(FILE *fin, MrcHeader *head, int x, int y, int z,
		void *pix_gray)

		{
	off_t offset;

	switch (head->mode)

	{
	case MRC_MODE_BYTE:
		offset = head->nx * head->ny;
		offset *= z;
		offset += (y * head->nx + x);
		offset *= sizeof(char);
		offset += 1024;
		fseeko(fin, offset, SEEK_SET);

		if ((fread(pix_gray, sizeof(char), 1, fin) == 0)) {
			printf(
					"Error with Function 'mrc_read_pixel()'! Reading file failed!");
			return false;
		}
		break;

	case MRC_MODE_SHORT:
		offset = head->nx * head->ny;
		offset *= z;
		offset += (y * head->nx + x);
		offset *= sizeof(short);
		offset += 1024;
		fseeko(fin, offset, SEEK_SET);
		//fseeko(fin,1024+(z*head->nx*head->ny+y*head->nx+x)*sizeof(short),SEEK_SET);

		if ((fread(pix_gray, sizeof(short), 1, fin) == 0)) {
			printf(
					"Error with Function 'mrc_read_pixel()'! Reading file failed!");
			return false;
		}
		break;

	case MRC_MODE_FLOAT:
		offset = head->nx * head->ny;
		offset *= z;
		offset += (y * head->nx + x);
		offset *= sizeof(float);
		offset += 1024;
		fseeko(fin, offset, SEEK_SET);
		//fseeko(fin,1024+(z*head->nx*head->ny+y*head->nx+x)*sizeof(float),SEEK_SET);

		if ((fread(pix_gray, sizeof(float), 1, fin) == 0)) {
			printf(
					"Error with Function 'mrc_read_pixel()'! Reading file failed!");
			return false;
		}
		break;

	}
	return 0;
}

/*******************************************************************************************/
int mrc_read_slice(FILE *fin, MrcHeader *head, int slcN, char axis,
		float *slcdata) {
//check the mrc file to make sure the size is exact in register with the head
	switch (head->mode) {
	case MRC_MODE_BYTE:
		if (get_file_size(fin) - (sizeof(MrcHeader) + head->next)
				!= head->nx * head->ny * head->nz * sizeof(char)) {
			printf("Error with Function 'mrc_read_slic()'! File size erro!\n");
		}
		break;

	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		if (get_file_size(fin) - (sizeof(MrcHeader) + head->next)
				!= (long int)head->nx * head->ny * head->nz * sizeof(short)) {
			printf("Error with Function 'mrc_read_slice()'! File size erro!\n");
		}
		break;

	case MRC_MODE_FLOAT:
		if (get_file_size(fin) - (sizeof(MrcHeader) + head->next)
				!= (long int)head->nx * head->ny * head->nz * sizeof(float)) {
			long int total_size = get_file_size(fin);
			int headsize = sizeof(MrcHeader) + head->next;
			printf("remainder:%lld shouldbe:%lld\n",get_file_size(fin) - (sizeof(MrcHeader) + head->next), (long long int)head->nx * head->ny * head->nz * sizeof(float));
			printf("%d %d %d %d \n", head->nx, head->ny, head->nz, head->next);
			fflush(stdout);
			printf("Error with Function 'mrc_read_slice()'! File size erro!\n");
		}
		break;

	default:
		printf("Error with Function 'mrc_read_slice()'! File type unknown!\n");

		break;
	}

	size_t psize;
	//int psize;
	short buf_short;
	unsigned short buf_ushort;
	unsigned char buf_byte;
	float buf_float;
	int i, k;

	switch (head->mode) {
	case MRC_MODE_BYTE:
		psize = sizeof(unsigned char);

		break;

	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		psize = sizeof(short);

		break;

	case MRC_MODE_FLOAT:
		psize = sizeof(float);

		break;
	}

	switch (axis) {

	/***********************************X************************************/
	case 'x':
	case 'X':

		fseek(fin, (sizeof(MrcHeader) + head->next) + slcN * psize, SEEK_SET);

		switch (head->mode) {
		case MRC_MODE_BYTE:
			for (i = 0; i < head->ny * head->nz; i++) {
				fread(&buf_byte, psize, 1, fin);
				slcdata[i] = (float) buf_byte;
				fseek(fin, (head->nx - 1) * psize, SEEK_CUR);
			}

			break;

		case MRC_MODE_SHORT:
			for (i = 0; i < head->ny * head->nz; i++) {
				fread(&buf_short, psize, 1, fin);
				slcdata[i] = (float) (buf_short);
				fseek(fin, (head->nx - 1) * psize, SEEK_CUR);
			}

			break;

		case MRC_MODE_USHORT:
			for (i = 0; i < head->ny * head->nz; i++) {
				fread(&buf_ushort, psize, 1, fin);
				slcdata[i] = (float) (buf_ushort);
				fseek(fin, (head->nx - 1) * psize, SEEK_CUR);
			}

			break;

		case MRC_MODE_FLOAT:
			for (i = 0; i < head->ny * head->nz; i++) {
				fread(&buf_float, psize, 1, fin);
				slcdata[i] = buf_float;
				fseek(fin, (head->nx - 1) * psize, SEEK_CUR);
			}
			break;

		}

		break;

		/***********************************Y************************************/
	case 'y':
	case 'Y':

		for (k = 0; k < head->nz; k++) {
			fseek(fin,
					(sizeof(MrcHeader) + head->next)
							+ (long int)psize
									* (k * (long int)head->nx * head->ny + head->nx * slcN),
					SEEK_SET);

			switch (head->mode) {
			case MRC_MODE_BYTE:
				for (i = 0; i < head->nx; i++) {
					fread(&buf_byte, psize, 1, fin);
					slcdata[k * head->nx + i] = (float) buf_byte;
				}

				break;

			case MRC_MODE_SHORT:
				for (i = 0; i < head->nx; i++) {
					fread(&buf_short, psize, 1, fin);
					slcdata[k * head->nx + i] = (float) (buf_short);
				}

				break;

			case MRC_MODE_USHORT:
				for (i = 0; i < head->nx; i++) {
					fread(&buf_ushort, psize, 1, fin);
					slcdata[k * head->nx + i] = (float) (buf_ushort);
				}

				break;

			case MRC_MODE_FLOAT:
				for (i = 0; i < head->nx; i++) {
					fread(&buf_float, psize, 1, fin);
					slcdata[k * head->nx + i] = buf_float;
				}

				break;

			}

		}
		break;

		/***********************************Z************************************/
	case 'z':
	case 'Z':
		fseek(fin,
				(sizeof(MrcHeader) + head->next)
						+ psize * slcN * head->nx * head->ny, SEEK_SET);

		if (head->mode == MRC_MODE_FLOAT)
			fread(slcdata, psize * head->nx * head->ny, 1, fin);

		else if (head->mode == MRC_MODE_BYTE) {
			for (i = 0; i < head->nx * head->ny; i++) {
				fread(&buf_byte, psize, 1, fin);
				slcdata[i] = (float) buf_byte;
			}
		}

		else if (head->mode == MRC_MODE_SHORT) {
			for (i = 0; i < head->nx * head->ny; i++) {
				fread(&buf_short, psize, 1, fin);
				slcdata[i] = (float) buf_short;
			}
		}

		else if (head->mode == MRC_MODE_USHORT) {
			for (i = 0; i < head->nx * head->ny; i++) {
				fread(&buf_ushort, psize, 1, fin);
				slcdata[i] = (float) buf_ushort;
			}
		}

		break;

	}

	return 0;
}
void mrc_read_slice_y(FILE *fin, MrcHeader *head, int slcN, float *slcdata) {
//check the mrc file to make sure the size is exact in register with the head
	int start =1024;
	switch (head->mode) {
	case MRC_MODE_BYTE:
		if (get_file_size(fin) - (sizeof(MrcHeader) + head->next)
				!= (long)head->nx * head->ny * head->nz * sizeof(char)) {
			printf("BYTE:Error with Function 'mrc_read_slic()'! File size erro!\n");
			start=get_file_size(fin) -head->nx * head->ny * head->nz * sizeof(char);
		}
		break;

	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		if (get_file_size(fin) - (sizeof(MrcHeader) + head->next)
				!= (long)head->nx * head->ny * head->nz * sizeof(short)) {
			printf("SHORT:Error with Function 'mrc_read_slice()'! File size erro!\n");
			start=get_file_size(fin)-head->nx * head->ny * head->nz * sizeof(short);
		}
		break;

	case MRC_MODE_FLOAT:
		if (get_file_size(fin) - (sizeof(MrcHeader) + head->next)
				!= (long)head->nx * head->ny * head->nz * sizeof(float)) {
			printf("FLOAT:Error with Function 'mrc_read_slice()'! File size erro!\n");
			start=get_file_size(fin) -head->nx * head->ny * head->nz * sizeof(float);
		}
		break;

	default:
		printf("Error with Function 'mrc_read_slice()'! File type unknown!\n");

		break;
	}

	size_t psize;
	//int psize;
	short buf_short;
	unsigned short buf_ushort;
	unsigned char buf_byte;
	float buf_float;
	int i, k;

	switch (head->mode) {
	case MRC_MODE_BYTE:
		psize = sizeof(unsigned char);
		break;
	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		psize = sizeof(short);
		break;
	case MRC_MODE_FLOAT:
		psize = sizeof(float);
		break;
	}

	for (k = 0; k < head->nz; k++) {
		fseek(fin, (sizeof(MrcHeader) + head->next) + psize * (k * head->nx * head->ny + head->nx * slcN),
				SEEK_SET);

		switch (head->mode) {
		case MRC_MODE_BYTE:
			//printf("MRC_MODE_BYTE\n");
			for (i = 0; i < head->nx; i++) {
				fread(&buf_byte, psize, 1, fin);
				slcdata[k * head->nx + i] = (float) buf_byte;
			}
			break;
		case MRC_MODE_SHORT:
			//printf("MRC_MODE_SHORT\n");
			for (i = 0; i < head->nx; i++) {
				fread(&buf_short, psize, 1, fin);
				slcdata[k * head->nx + i] = (float) (buf_short);
			}
			break;
		case MRC_MODE_USHORT:
			//printf("MRC_MODE_USHORT\n");
			for (i = 0; i < head->nx; i++) {
				fread(&buf_ushort, psize, 1, fin);
				slcdata[k * head->nx + i] = (float) (buf_ushort);
			}
			break;
		case MRC_MODE_FLOAT:
			//printf("MRC_MODE_FLOAT\n");
			for (i = 0; i < head->nx; i++) {
				fread(&buf_float, psize, 1, fin);
				slcdata[k * head->nx + i] = buf_float;
			}
			break;
		}

	}
}
/*****************************************************************************************************/

int mrc_read_block(FILE *fin, MrcHeader *head, int start, int end, char axis,
		float *mrc_data_block) {

//check the mrc file to make sure the size is exact in register with the head
	switch (head->mode) {
	case MRC_MODE_BYTE:
		if (get_file_size(fin) - 1024
				!= head->nx * head->ny * head->nz * sizeof(char)) {
			printf("Error with Function 'mrc_read_block()'! File size erro!");
		}
		break;

	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		if (get_file_size(fin) - 1024
				!= head->nx * head->ny * head->nz * sizeof(short)) {
			printf("Error with Function 'mrc_read_block()'! File size erro!");
		}
		break;

	case MRC_MODE_FLOAT:
		if (get_file_size(fin) - 1024
				!= head->nx * head->ny * head->nz * sizeof(float)) {
			printf("Error with Function 'mrc_read_block()'! File size erro!");
		}
		break;

	default:
		printf("Error with Function 'mrc_read_block()'! File type unknown!");

		break;
	}

	int i, k;
	unsigned char buf_byte;
	short buf_short;
	short buf_ushort;

	int psize;

	off_t offset;
	switch (head->mode) {
	case MRC_MODE_BYTE:
		psize = sizeof(unsigned char);

		break;

	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		psize = sizeof(short);

		break;

	case MRC_MODE_FLOAT:
		psize = sizeof(float);

		break;
	}

	switch (axis) {

	/***********************************X************************************/
	case 'x':
	case 'X':

		/*fseeko(fin,1024+slcN*psize,SEEK_SET);

		 switch(head->mode)
		 {
		 case MRC_MODE_BYTE:
		 for(i=0;i<head->ny*head->nz;i++)
		 {
		 fread(&buf_byte,psize,1,fin);
		 slcdata[i]=(float)buf_byte;
		 fseeko(fin,(head->nx-1)*psize,SEEK_CUR);
		 }

		 break;

		 case MRC_MODE_SHORT:
		 for(i=0;i<head->ny*head->nz;i++)
		 {
		 fread(&buf_short,psize,1,fin);
		 slcdata[i]=(float)(buf_short);
		 fseeko(fin,(head->nx-1)*psize,SEEK_CUR);
		 }

		 break;

		 case MRC_MODE_FLOAT:
		 for(i=0;i<head->ny*head->nz;i++)
		 {
		 fread(&buf_float,psize,1,fin);
		 slcdata[i]=buf_float;
		 fseeko(fin,(head->nx-1)*psize,SEEK_CUR);
		 }
		 break;

		 }*/

		break;

		/***********************************Y************************************/
	case 'y':
	case 'Y':
		//   fseeko(fin,1024+(k*head->nx*head->ny+head->nx*slcN)*psize,SEEK_SET);
		/* fseeko(fin,1024+slcN*head->nx*psize,SEEK_SET);
		 for(k=0;k<head->nz;k++)
		 {

		 switch(head->mode)
		 {
		 case MRC_MODE_BYTE:
		 for(i=0;i<head->nx;i++)
		 {
		 fread(&buf_byte,psize,1,fin);
		 slcdata[k*head->nx+i]=(float)buf_byte;
		 }

		 break;

		 case MRC_MODE_SHORT:
		 for(i=0;i<head->nx;i++)
		 {
		 fread(&buf_short,psize,1,fin);
		 slcdata[k*head->nx+i]=(float)(buf_short);
		 }

		 break;

		 case MRC_MODE_FLOAT:
		 fread(slcdata+k*head->nx,psize,head->nx,fin);


		 break;

		 }//end switch
		 fseeko(fin,head->nx*(head->ny-1)*psize,SEEK_CUR);
		 }//end for*/
		break;

		/***********************************Z************************************/
	case 'z':
	case 'Z':
		offset = head->nx * head->ny;
		offset *= (start * psize);
		offset += (1024);
		fseeko(fin, offset, SEEK_SET);
		//fseeko(fin,1024+slcN*head->nx*head->ny*psize,SEEK_SET);

		switch (head->mode) {
		case MRC_MODE_BYTE:
			for (i = 0; i < head->nx * head->ny * (end - start); i++) {
				fread(&buf_byte, psize, 1, fin);
				mrc_data_block[i] = (float) buf_byte;
			}
			break;

		case MRC_MODE_SHORT:
		case MRC_MODE_USHORT:
			for (i = 0; i < head->nx * head->ny * (end - start); i++) {
				fread(&buf_short, psize, 1, fin);
				mrc_data_block[i] = (float) buf_short;
			}
			break;

		case MRC_MODE_FLOAT:
			fread(mrc_data_block, psize, head->nx * head->ny * (end - start),
					fin);
			break;
		}

		break;

	}
	return 0;
}

/*****************************************************************************************************/

int mrc_read_all(FILE *fin, MrcHeader *head, float *mrc_data_all) {
	int headSize = 1024;
//check the mrc file to make sure the size is exact in register with the head
	switch (head->mode) {
	case MRC_MODE_BYTE:
		/*if (get_file_size(fin) - 2718
				!= head->nx * head->ny * head->nz * sizeof(char)) {
			printf("1.Error with Function 'mrc_read_all()'! File size erro!");
		}*/
		headSize = get_file_size(fin)
						- head->nx * head->ny * head->nz * sizeof(char);
		break;

	case MRC_MODE_SHORT:
	case MRC_MODE_USHORT:
		/*if (get_file_size(fin) - 2718
				!= head->nx * head->ny * head->nz * sizeof(short)) {
			printf("2.Error with Function 'mrc_read_all()'! File size erro!");
		}*/
		headSize = get_file_size(fin)
								- head->nx * head->ny * head->nz * sizeof(short);
		break;

	case MRC_MODE_FLOAT:
		/*if(get_file_size(fin) - 2718 != head->nx*head->ny*head->nz*sizeof(float))
		 {
		 printf("3.Error with Function 'mrc_read_all()'! File size erro!");
		 }
		 */
		headSize = get_file_size(fin)
				- head->nx * head->ny * head->nz * sizeof(float);
		//printf("%d",headSize);
		break;

	default:
		printf("4.Error with Function 'mrc_read_all()'! File type unknown!");

		break;
	}

	long i;
	unsigned char buf_byte;
	short buf_short;
	short buf_ushort;

	fseek(fin, (headSize), SEEK_SET);

	switch (head->mode) {
	case MRC_MODE_BYTE:

		for (i = 0; i < head->nx * head->ny * head->nz; i++) {
			fread(&buf_byte, sizeof(char), 1, fin);
			mrc_data_all[i] = (float) buf_byte;
		}
		break;

	case MRC_MODE_SHORT:
		for (i = 0; i < head->nx * head->ny * head->nz; i++) {
			fread(&buf_short, sizeof(short), 1, fin);
			mrc_data_all[i] = (float) buf_short;
		}
		break;

	case MRC_MODE_USHORT:

		for (i = 0; i < head->nx * head->ny * head->nz; i++) {
			fread(&buf_ushort, sizeof(short), 1, fin);
			mrc_data_all[i] = (float) buf_ushort;
		}
		break;

	case MRC_MODE_FLOAT:

		if ((fread(mrc_data_all, head->nx * head->ny * head->nz * sizeof(float),
				1, fin) == 0)) {
			printf(
					"Error with Function 'mrc_read_all()'! Reading file failed!");
			return -1;
		}
		break;

	default:
		printf("Error with Function 'mrc_read_all()'! File type unknown!");
		break;

	}

	return 0;
}

/*******************************************************************************************/
int mrc_write_slice(FILE *fout, MrcHeader *head, int slcN, char axis,
		float *slcdata) {
	int psize;
	if (head->mode == MRC_MODE_FLOAT)
		psize = sizeof(float);
	else {
		head->mode = MRC_MODE_FLOAT;
		printf("outfile headmode is error!\n");
		return false;
	}

	int i, k;
	off_t offset;

	switch (axis) {

	/***********************************X************************************/
	case 'x':
	case 'X':

		fseeko(fout, 1024 + slcN * psize, SEEK_SET);

		for (i = 0; i < head->ny * head->nz; i++) {
			fwrite(slcdata + i, psize, 1, fout);
			fseeko(fout, (head->nx - 1) * psize, SEEK_CUR);
		}
		break;

		/***********************************Y************************************/
	case 'y':
	case 'Y':
		fseeko(fout, 1024 + slcN * head->nx * psize, SEEK_SET);

		for (k = 0; k < head->nz; k++) {
			fwrite(slcdata + k * head->nx, psize, head->nx, fout);
			fseeko(fout, head->nx * (head->ny - 1) * psize, SEEK_CUR);
		}
		break;

		/***********************************Z************************************/
	case 'z':
	case 'Z': //problem
		//fseeko(fout,1024+slcN*head->nx*head->ny*psize,SEEK_SET);
		offset = head->nx * head->ny;
		offset *= (slcN * psize);
		offset += 1024;
		fseeko(fout, offset, SEEK_SET);
		fwrite(slcdata, psize, head->nx * head->ny, fout);

		break;

	}
	return 0;
}

/*******************************************************************************************/
int mrc_add_sliceN(FILE *fout, MrcHeader *headout, float *slcdata, int slcN) {
	off_t length;
	//fseeko(fout,1024+sizeof(float)*headout->nx*headout->ny*slcN,SEEK_SET);
	length = headout->nx * headout->ny;
	length *= (sizeof(float) * slcN);
	length += 1024;
	fseeko(fout, length, SEEK_SET);

	fwrite(slcdata, sizeof(float), headout->nx * headout->ny, fout);
	return 0;
}

/*******************************************************************************************/
int mrc_add_slice(FILE *fout, MrcHeader *headout, float *slcdata) {

	fseeko(fout, 0, SEEK_END);
	fwrite(slcdata, sizeof(float), headout->nx * headout->ny, fout);
	return 0;
}

/*****************************************************************************************************/

int mrc_write_all(FILE *fout, MrcHeader *head, int Z_start, int Z_end,
		void *mrc_data_all) {
	int psize;

	switch (head->mode)

	{
	case MRC_MODE_BYTE:
		psize = sizeof(char);

		break;

	case MRC_MODE_SHORT:
		psize = sizeof(short);

		break;

	case MRC_MODE_FLOAT:
		psize = sizeof(float);

		break;
	}

	off_t offset;
	offset = head->nx * head->ny;
	offset *= (Z_start * psize);
	offset += 1024;
	fseeko(fout, offset, SEEK_SET);

	if ((fwrite(mrc_data_all, psize, head->nx * head->ny * (Z_end - Z_start),
			fout) == 0)) {
		printf("Error with Function 'mrc_read_all()'! Reading file failed!");
		return false;
	}

	return true;
}

/*******************************************************************************************/
/*int mrc_flipyz(char *inf,char *outf,int myid,int mypro)
 {
 printf("\nBegin flipping:");
 MrcHeader  *inhead,*outhead;
 inhead=(MrcHeader *)malloc(sizeof(MrcHeader));
 outhead=(MrcHeader *)malloc(sizeof(MrcHeader));

 FILE *fin,*fout;
 if((fin=fopen(inf,"r"))==NULL)
 {
 printf("\nCannot open file strike any key exit!");
 return false;
 }

 if((fout=fopen(outf,"w+"))==NULL)
 {
 printf("\nCannot open file strike any key exit!");
 return false;
 }

 mrc_read_head(fin,inhead);
 mrc_read_head(fin,outhead);

 outhead->nx=inhead->nx;
 outhead->ny=inhead->nz;
 outhead->nz=inhead->ny;
 outhead->mode=MRC_MODE_FLOAT;
 mrc_write_head(fout,outhead);
 printf("inhead.ny is %d\n",inhead->ny);

 int start_process;   //the start slice of reproject per process
 int end_process  ;   //the end slice of reproject per process
 start_process=myid*inhead->ny/mypro;
 end_process  =(myid+1)*inhead->ny/mypro;

 int sliceYZ=0;
 int global_sliceYZ=0;

 float *buf;
 buf=(float *)malloc(sizeof(float)*inhead->nx*inhead->nz);
 int j;

 for(j=start_process;j<end_process;j++)
 {
 mrc_read_slice(fin,inhead,j,'y',buf);
 //mrc_add_slice(fout,outhead,buf);
 mrc_add_sliceN(fout,outhead,buf,j);
 printf("%d of %d slice flipping finished  of process %d\n",j+1,inhead->ny,myid);
 sliceYZ++;
 }

 free (buf);
 printf("sliceYZ number in mrc_flipyz is %d in process %d\n",sliceYZ,myid);

 fclose(fin);
 fclose(fout);
 free (inhead);
 free (outhead);
 MPI_Reduce(&sliceYZ,&global_sliceYZ,1,MPI_INT,MPI_SUM,0,MPI_COMM_WORLD);
 if(!myid)
 {
 printf("%d slice is flipped!\n",global_sliceYZ);
 printf("\nflipping finished!\n");
 }
 return 0;
 }

 */

/*******************************************************************************************/
int mrc_upsidedown(char *inf, char *outf) {
	MrcHeader *inhead, *outhead;
	inhead = (MrcHeader *) malloc(sizeof(MrcHeader));
	outhead = (MrcHeader *) malloc(sizeof(MrcHeader));

	FILE *fin, *fout;
	if ((fin = fopen(inf, "r")) == NULL) {
		printf("\nCannot open file strike any key exit!");
	}

	if ((fout = fopen(outf, "w+")) == NULL) {
		printf("\nCannot open file strike any key exit!");
	}

	mrc_read_head(fin, inhead);
	mrc_read_head(fin, outhead);

	outhead->mode = MRC_MODE_FLOAT;
	mrc_write_head(fout, outhead);

	float *buf;
	buf = (float *) malloc(sizeof(float) * inhead->nx * inhead->ny);
	int j;
	for (j = 0; j < inhead->nz; j++) {
		mrc_read_slice(fin, inhead, inhead->nz - 1 - j, 'z', buf);
		mrc_add_slice(fout, outhead, buf);
	}
	free(buf);
	free(inhead);
	free(outhead);
	fclose(fin);
	fclose(fout);
	return 0;
}

/****************************************************************************************/
/*
 void mrc_rotx(char *inf,char *outf,float ang,int thick)
 {
 printf("\nBegin rotating:");
 FILE *fin,*fout;
 if((fin=fopen64(inf,"r"))==NULL)
 {
 printf("\nCannot open file strike any key exit!");
 }

 MrcHeader  *inhead,*outhead;
 inhead=(MrcHeader *)malloc(sizeof(MrcHeader));
 outhead=(MrcHeader *)malloc(sizeof(MrcHeader));
 mrc_read_head(fin,inhead);
 mrc_read_head(fin,outhead);

 float *indata,*outdata;
 indata=(float *)malloc(sizeof(float)*inhead->ny*inhead->nz);
 outdata=(float *)malloc(sizeof(float)*inhead->ny*thick);
 int yp,zp,yp0,zp0;
 int y,z,y0,z0;
 yp0=inhead->ny/2;
 zp0=inhead->nz/2;
 y0=yp0;
 z0=thick/2;


 outhead->nz=thick;

 mrc_new(outf,outhead);

 if((fout=fopen64(outf,"a+"))==NULL)
 {
 printf("\nCannot open file strike any key exit!");
 }
 int i,j,k,r;
 for(i=0;i<inhead->nx;i++)
 {
 mrc_read_slice(fin,i,'x',indata);
 for(r=0;r<inhead->ny*thick;i++)
 {
 y=r%thick; z=r/thick;
 yp=int((y-y0)*cos(ang)-(z-z0)*sin(ang)+yp0);
 zp=int((y-y0)*sin(ang)+(z-z0)*cos(ang)+zp0);
 j=zp*inhead->ny+yp;
 if(j>=0&&j<inhead->ny*inhead->nz)outdata[r]=indata[j];
 else outdata[r]=0;
 }
 mrc_write_slcX(fout,i,outdata);
 printf("\n%d of %d slice finished");
 }
 fclose(fout);
 printf("\nRotating finished!");

 }
 */

/*int main(int argc, char *argv[])
 {
 mrc_flipyz(argv[1],argv[2]);
 }*/

/****************************************************************************************/
/*int main(int argc,char *argv[])
 {
 char *inf="tempflip.rec";
 char *outf="cavfinal.rec";
 float ang=rad(11.62);
 int thick=100;
 printf("rotating:\n");
 FILE *fin,*fout;
 if((fin=fopen64(inf,"r"))==NULL)
 {
 printf("\nCannot open file strike any key exit!");
 }

 MrcHeader  *inhead,*outhead;
 inhead=(MrcHeader *)malloc(sizeof(MrcHeader));
 outhead=(MrcHeader *)malloc(sizeof(MrcHeader));
 mrc_read_head(fin,inhead);
 mrc_read_head(fin,outhead);

 float *indata,*outdata;
 indata=(float *)malloc(sizeof(float)*inhead->ny*inhead->nz);
 //outdata=(float *)malloc(sizeof(float)*inhead->ny*inhead->nz);
 outdata=(float *)malloc(sizeof(float)*inhead->ny*thick);
 int yp,zp,yp0,zp0;
 int y,z,y0,z0;
 yp0=inhead->ny/2;
 zp0=inhead->nz/2;
 y0=yp0;
 //z0=inhead->nz/2;
 z0=thick/2;

 // outhead->nx=inhead->ny;
 outhead->nz=thick;

 // outhead->ny=inhead->nz;

 mrc_new(outf,outhead);

 if((fout=fopen64(outf,"r+"))==NULL)
 {
 printf("\nCannot open file strike any key exit!");
 }
 int i,j,k,r;
 printf("ok1");
 for(i=0;i<inhead->nx;i++)
 //for(i=0;i<3;i++)
 {
 mrc_read_slice(fin,i,'x',indata);
 for(r=0;r<inhead->ny*thick;r++)
 //for(r=0;r<inhead->ny*inhead->nz;r++)
 { //j=r+10*inhead->ny;
 //j=r;
 y=r%inhead->ny; z=r/inhead->ny;
 yp=int((y-y0)*cos(ang)-(z-z0)*sin(ang)+yp0);
 zp=int((y-y0)*sin(ang)+(z-z0)*cos(ang)+zp0);
 j=zp*inhead->ny+yp;
 if(j>=0&&j<inhead->ny*inhead->nz)outdata[r]=indata[j];
 else outdata[r]=0;
 }
 mrc_write_slcX(fout,i,outdata);
 printf("\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b");
 printf("%4d of %4d slice finished",i,inhead->nx);

 }

 fclose(fout);
 printf("\nRotating finished!\n");
 }*/
