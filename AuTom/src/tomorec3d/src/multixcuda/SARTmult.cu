#include "hip/hip_runtime.h"
#include "SARTmult.cuh"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
__constant__ ImageStruct dev_const_img;
__constant__ Volume dev_const_vol;
__constant__ Projection dev_const_prj;
__constant__ Pixel0 dev_const_Pixel0;

long int initGPU(int mpi_rank_id) {

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int gpu_id;
#ifdef ICT_NORMAL
	gpu_id = mpi_rank_id % deviceCount;
#else
	if (mpi_rank_id == 0) {
		gpu_id = 0;
	} else {
		gpu_id = 2;
	}

#endif
	hipSetDevice(gpu_id);
	hipDeviceReset();
	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, gpu_id);

	printf("mpi=%d GPU=%d device properties is :\n"
			"\t device name is %s\n"
			"\t totalGlobalMem is %d MB\n"
			"\t sharedMemPerBlock is %d\n"
			"\t regsPerBlock is %d\n"
			"\t memPitch is %d\n"
			"\t maxThreadsPerBlock is %d\n"
			"\t maxThreadsDim [3] is %d X %d X %d\n"
			"\t maxGridSize [3] is %d X %d X %d\n"
			"\t totalConstMem is %d\n"
			"\t device version is major %d ,minor %d\n"
			"\t clockRate is %d\n"
			"\t textureAlignment is %d\n"
			"\t deviceOverlap is %d\n"
			"\t multiProcessorCount is %d\n", mpi_rank_id, gpu_id,
			device_prop.name, device_prop.totalGlobalMem / 1024 / 1024,
			device_prop.sharedMemPerBlock, device_prop.regsPerBlock,
			device_prop.memPitch, device_prop.maxThreadsPerBlock,
			device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1],
			device_prop.maxThreadsDim[2], device_prop.maxGridSize[0],
			device_prop.maxGridSize[1], device_prop.maxGridSize[2],
			device_prop.totalConstMem, device_prop.major, device_prop.minor,
			device_prop.clockRate, device_prop.textureAlignment,
			device_prop.deviceOverlap, device_prop.multiProcessorCount);
	hipDeviceReset();
	return device_prop.totalGlobalMem;
}

int initMalloc(ImageStruct & host_img, int prj_AngN, int line_num,
		long int total_mem) {
	long int total_used_mem = 0;

	//CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img_ptr, sizeof(ImageStruct)));
	total_used_mem += sizeof(ImageStruct);

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.x_coef,
					sizeof(double) * prj_AngN * 3));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.y_coef,
					sizeof(double) * prj_AngN * 3));
	total_used_mem += sizeof(double) * prj_AngN * 6;

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_calc, sizeof(float) * line_num));

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_real, sizeof(float) * line_num));

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_calc_num,
					sizeof(float) * line_num));


	total_used_mem += (sizeof(float) * 5 + sizeof(double) * 2) * line_num;


	long int mem = total_mem - total_used_mem;
	int total_z = mem / (host_img.vol.X * host_img.vol.Y * sizeof(float));
	printf("total_z %d\n",total_z);
	int count = 0;
	if (total_z > host_img.vol.volZ_per) {
		total_z = host_img.vol.volZ_per;
		count = 1;
	} else {
		count = (host_img.vol.volZ_per - 1) / total_z + 1;
		total_z = (host_img.vol.volZ_per - 1) / count + 1;
	}
	host_img.vol.Zcount = count;
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.slc_data,
					host_img.vol.X * host_img.vol.Y * sizeof(float) * total_z));

	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &host_img, sizeof(ImageStruct));
	return count;

}
int ASARTinitMalloc(ImageStruct & host_img, int prj_AngN, int line_num,
		long int total_mem) {
	long int total_used_mem = 0;

	//CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img_ptr, sizeof(ImageStruct)));
	total_used_mem += sizeof(ImageStruct);

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.x_coef,
					sizeof(double) * prj_AngN * 3));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.y_coef,
					sizeof(double) * prj_AngN * 3));
	total_used_mem += sizeof(double) * prj_AngN * 6;

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_calc, sizeof(float) * line_num));

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_real, sizeof(float) * line_num));

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_calc_num,
					sizeof(float) * line_num));

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.prj_calc_abs,
					sizeof(float) * line_num));


	total_used_mem += (sizeof(float) * 5 + sizeof(double) * 2) * line_num;


	long int mem = total_mem - total_used_mem;
	int total_z = mem / (host_img.vol.X * host_img.vol.Y * sizeof(float));
	printf("total_z %d\n",total_z);
	int count = 0;
	if (total_z > host_img.vol.volZ_per) {
		total_z = host_img.vol.volZ_per;
		count = 1;
	} else {
		count = (host_img.vol.volZ_per - 1) / total_z + 1;
		total_z = (host_img.vol.volZ_per - 1) / count + 1;
	}
	host_img.vol.Zcount = count;
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&host_img.slc_data,
					host_img.vol.X * host_img.vol.Y * sizeof(float) * total_z));

	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &host_img, sizeof(ImageStruct));
	return count;

}
void initDevMem(ImageStruct host_img, int line_num) {

	CUDA_CHECK_RETURN(
			hipMemset(host_img.prj_calc, 0, sizeof(float) * line_num));
	CUDA_CHECK_RETURN(
			hipMemset(host_img.prj_real, 0, sizeof(float) * line_num));
	CUDA_CHECK_RETURN(
			hipMemset(host_img.prj_calc_num, 0, sizeof(float) * line_num));

}
//init prj_calc & prj_calc_num
void initprj_calcMem(float *prj_calc, int line_num) {

	CUDA_CHECK_RETURN(hipMemset(prj_calc, 0, sizeof(float) * line_num));
}
void transferToDev(ImageStruct *dev_img_ptr, ImageStruct *host_img_ptr,
		int prj_AngN) {

	CUDA_CHECK_RETURN(
			hipMemcpy(dev_img_ptr->x_coef, host_img_ptr->x_coef,
					sizeof(double) * prj_AngN * 3, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_img_ptr->y_coef, host_img_ptr->y_coef,
					sizeof(double) * prj_AngN * 3, hipMemcpyHostToDevice));
}
void transferToDevPro(float *dev_prj_real, float *prj_real, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_prj_real, prj_real, size, hipMemcpyHostToDevice));
}
void transferToHostPro(float *dev_prj_real, float *prj_real, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(prj_real, dev_prj_real, size, hipMemcpyDeviceToHost));
}
void transferToDevVol(float *dev_slc_data, float *slc_data, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_slc_data, slc_data, size, hipMemcpyHostToDevice));
}
void transferToHost(float *slc_data, float *dev_slc_data, int pixel_num_per) {
	CUDA_CHECK_RETURN(
			hipMemcpy(slc_data, dev_slc_data, pixel_num_per,
					hipMemcpyDeviceToHost));
}
//////FBP
void initSlcMalloc(float *&slc_data, int pixel_num) {
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&slc_data, sizeof(float) * pixel_num));
}
void initSlcDevMem(float *slc_data, int pixel_num) {

	CUDA_CHECK_RETURN(hipMemset(slc_data, 0, sizeof(float) * pixel_num));
}
void initProjectionMalloc(float *&prj, int line_num) {
	CUDA_CHECK_RETURN(hipMalloc((void ** )&prj, sizeof(float) * line_num));
}
void transferProjectionToDev(float *dev_prj, float *host_prj, int line_num) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_prj, host_prj, sizeof(float) * line_num,
					hipMemcpyHostToDevice));
}
void initMalloc_FBP(double *&x_coef, double *&y_coef,
		int prj_AngN) {
	//CUDA_CHECK_RETURN(hipMalloc((void ** )&Pix10, sizeof(Pixel0)));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&x_coef, sizeof(double) * prj_AngN * 3));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&y_coef, sizeof(double) * prj_AngN * 3));
}
void initDevMem_FBP(double *x_coef, double *y_coef, int prj_AngN) {

	//CUDA_CHECK_RETURN(hipMemset(Pix10, 0, sizeof(Pixel0)));
	CUDA_CHECK_RETURN(hipMemset(x_coef, 0, sizeof(double) * prj_AngN * 3));
	CUDA_CHECK_RETURN(hipMemset(y_coef, 0, sizeof(double) * prj_AngN * 3));
}

void transferToDev_FBP(Pixel0 *host_Pixel0, double *dev_x_coef, double *host_x_coef,
		double *dev_y_coef, double *host_y_coef, int prj_AngN,
		Projection host_prj, Volume host_volume) {

	hipMemcpyToSymbol((const void*) &dev_const_Pixel0, (void*) host_Pixel0,
			sizeof(Pixel0));
	hipMemcpyToSymbol((const void*) &dev_const_prj, (void*) &host_prj,
			sizeof(Projection));
	hipMemcpyToSymbol((const void*) &dev_const_vol, (void*) &host_volume,
			sizeof(Volume));

	CUDA_CHECK_RETURN(
			hipMemcpy(dev_x_coef, host_x_coef, sizeof(double) * prj_AngN * 3,
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_y_coef, host_y_coef, sizeof(double) * prj_AngN * 3,
					hipMemcpyHostToDevice));
}
__device__ void FBP_Projection(int x, int y, float x_weight, float y_weight,
		int n, float *prj_real, int prjx, float *slc_data, int prjAngN) {
	int m = x + y * prjx; //prj index
	atomicAdd(&slc_data[n], (x_weight * y_weight * prj_real[m]) / prjAngN);
//	if(blockIdx.y==1&&blockIdx.x==0&&threadIdx.x==0){
//		printf("id =%d %d slc=%f prj=%f \n", blockIdx.x, threadIdx.x,slc_data[n],   prj_real[m]);
//	}
}
__device__ void dev_val_coef(Pixel0 *pixel0, Pixel pixel, int angle,
		float zshift, double *x_coef, double *y_coef, Weight *comp_prj)

		{
	double x, y;
	int index;
	index = 3 * angle;  //order=1,straight model

	x = x_coef[index] * (pixel.X - pixel0->X)
			+ x_coef[index + 1] * (pixel.Y - pixel0->Y)
			+ x_coef[index + 2] * (pixel.Z - pixel0->Z - zshift) + pixel0->X;
	y = y_coef[index] * (pixel.X - pixel0->X)
			+ y_coef[index + 1] * (pixel.Y - pixel0->Y)
			+ y_coef[index + 2] * (pixel.Z - pixel0->Z - zshift) + pixel0->Y;

	comp_prj->x_min = floor(x);
	comp_prj->y_min = floor(y);

	comp_prj->x_min_del = x - comp_prj->x_min;
	comp_prj->y_min_del = y - comp_prj->y_min;

}
__global__ void FBP_Slice_prj(int total_num_slc, int z_start, float *prj_real,
		float *slc_data, int zshift, double *dev_x_coef,
		double *dev_y_coef,int prjoffset,int voloffset) {

	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;
	int angindex=blockIdx.y+prjoffset;
	//printf("GPU info --%d  total :%d \n",n,total_num_slc);
	if (n < total_num_slc) {
		//printf("GPU info --%d  total :%d \n",n,total_num_slc);
		//get the x y z of volume
		Pixel pixel;
		Weight comp_prj;

		pixel.Z = (n) / (dev_const_vol.X * dev_const_vol.Y)
				+ z_start; //+vol_ptr->Zstart_per
		pixel.Y = (n) % (dev_const_vol.X * dev_const_vol.Y)
				/ dev_const_vol.X + dev_const_vol.Ystart;
		pixel.X = (n) % (dev_const_vol.X * dev_const_vol.Y)
				% dev_const_vol.X + dev_const_vol.Xstart;
		prj_real+=blockIdx.y*(dev_const_prj.X*dev_const_prj.Y);
		//get the x y of projection
		dev_val_coef(&dev_const_Pixel0, pixel, angindex, zshift, dev_x_coef,
				dev_y_coef, &comp_prj); //&dev_Pixel0
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_prj.X
				&& comp_prj.y_min >= 0 && comp_prj.y_min < dev_const_prj.Y) {

			FBP_Projection(comp_prj.x_min, comp_prj.y_min,
					1 - comp_prj.x_min_del, 1 - comp_prj.y_min_del, n, prj_real,
					dev_const_prj.X, slc_data, dev_const_prj.AngN);
		}
		if ((comp_prj.x_min + 1) >= 0 && (comp_prj.x_min + 1) < dev_const_prj.X
				&& comp_prj.y_min >= 0 && comp_prj.y_min < dev_const_prj.Y) //(x_min+1, y_min)
						{
			FBP_Projection(comp_prj.x_min + 1, comp_prj.y_min,
					comp_prj.x_min_del, 1 - comp_prj.y_min_del, n, prj_real,
					dev_const_prj.X, slc_data, dev_const_prj.AngN);
		}
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_prj.Y) //(x_min, y_min+1)
						{
			FBP_Projection(comp_prj.x_min, comp_prj.y_min + 1,
					1 - comp_prj.x_min_del, comp_prj.y_min_del, n, prj_real,
					dev_const_prj.X, slc_data, dev_const_prj.AngN);
		}
		if ((comp_prj.x_min + 1) >= 0 && (comp_prj.x_min + 1) < dev_const_prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_prj.Y) //(x_min+1, y_min+1)
						{
			FBP_Projection(comp_prj.x_min + 1, comp_prj.y_min + 1,
					comp_prj.x_min_del, comp_prj.y_min_del, n, prj_real,
					dev_const_prj.X, slc_data, dev_const_prj.AngN);
		}
	}
}
void Sart_Slice_backproj_bilinear(int pixel_num, int z_start, float *prj_real,
		float *slc_data, int zshift, double *dev_x_coef,
		double *dev_y_coef,int prjnum,int prjoffset,int voloffset) {
	int num_blocks = (pixel_num - 1) / BLOCKDIM + 1;
	dim3 dim2block(num_blocks,prjnum);
	//printf("pixel_num=%d prjnum=%d \n ", pixel_num, prjnum);
	FBP_Slice_prj<<<dim2block, BLOCKDIM>>>(pixel_num, z_start, prj_real,
			slc_data,  zshift, dev_x_coef, dev_y_coef,prjoffset,voloffset);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
}
void transferToHost_FBP(float *dev_slc_data, float *slc_data, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(slc_data, dev_slc_data, size * sizeof(float),
					hipMemcpyDeviceToHost));
}
void freeDevSlc(float *slc_data) {
	hipFree(slc_data);
}
void freeDevMem(double *x_coef,double *y_coef) {

	hipFree(x_coef);
	hipFree(y_coef);

}
void freeDevProjection(float *prj) {
	hipFree(prj);
}
//end BPT
void freeDevMem(ImageStruct *dev_img_ptr) {

	hipFree(dev_img_ptr->prj_calc);
	hipFree(dev_img_ptr->global_prj_calc);
	hipFree(dev_img_ptr->global_prj_calc_num);
	hipFree(dev_img_ptr->prj_calc_num);
	hipFree(dev_img_ptr->prj_real);
	hipFree(dev_img_ptr->slc_data);
	hipFree(dev_img_ptr->x_coef);
	hipFree(dev_img_ptr->y_coef);
}
__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}


__device__ void atomProjection(int x, int y, double x_weight, double y_weight,
		int n) {
	int prj_index;
	prj_index = x + y * dev_const_img.prj.X; //prj index
	double tmp = x_weight * y_weight * dev_const_img.slc_data[n];

	atomicAdd(&dev_const_img.prj_calc[prj_index], tmp);
	//dev_const_img.prj_calc[prj_index]+=tmp;
	atomicAdd(&dev_const_img.prj_calc_num[prj_index], x_weight * y_weight);

}
__device__ void ASARTatomProjection(int x, int y, double x_weight, double y_weight,
		int n) {
	int prj_index;
	prj_index = x + y * dev_const_img.prj.X; //prj index
	double tmp = x_weight * y_weight * dev_const_img.slc_data[n];
	double tmp2 = x_weight * y_weight * abs(dev_const_img.slc_data[n]);
	atomicAdd(&dev_const_img.prj_calc[prj_index], tmp);
	//dev_const_img.prj_calc[prj_index]+=tmp;
	atomicAdd(&dev_const_img.prj_calc_num[prj_index], x_weight * y_weight);
	atomicAdd(&dev_const_img.prj_calc_abs[prj_index], tmp2);

}
__global__ void SART_Slice_reproj_kernel(int ang, float zshift, int z_start,
		int total_num_slc) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;

	if (n < total_num_slc) {
		//get the x y z of volume
		Pixel pixel;
		Weight comp_prj;
		Volume *vol_ptr = &(dev_const_img.vol);
		pixel.Z = n / (vol_ptr->X * vol_ptr->Y) + vol_ptr->Zstart_per + z_start;
		pixel.Y = n % (vol_ptr->X * vol_ptr->Y) / vol_ptr->X + vol_ptr->Ystart;
		pixel.X = n % (vol_ptr->X * vol_ptr->Y) % vol_ptr->X + vol_ptr->Xstart;

		//get the x y of projection
		dev_val_coef(&dev_const_img.pixel0, pixel, ang, zshift,
				dev_const_img.x_coef, dev_const_img.y_coef, &comp_prj);

		//calculate the projection
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& comp_prj.y_min >= 0
				&& comp_prj.y_min < dev_const_img.prj.Y) {
			atomProjection(comp_prj.x_min, comp_prj.y_min,
					1 - comp_prj.x_min_del, 1 - comp_prj.y_min_del, n);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& comp_prj.y_min >= 0 && comp_prj.y_min < dev_const_img.prj.Y) //(x_min+1, y_min)
						{
			atomProjection(comp_prj.x_min + 1, comp_prj.y_min,
					comp_prj.x_min_del, 1 - comp_prj.y_min_del, n);
		}
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min, y_min+1)
						{
			atomProjection(comp_prj.x_min, comp_prj.y_min+1,
					1 - comp_prj.x_min_del, comp_prj.y_min_del, n);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min+1, y_min+1)
						{
			atomProjection(comp_prj.x_min + 1, comp_prj.y_min + 1,
					comp_prj.x_min_del, comp_prj.y_min_del, n);
		}

	} //end if(n<total_num_slc)
}
__global__ void ASART_Slice_reproj_kernel(int ang, float zshift, int z_start,
		int total_num_slc) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;

	if (n < total_num_slc) {
		//get the x y z of volume
		Pixel pixel;
		Weight comp_prj;
		Volume *vol_ptr = &(dev_const_img.vol);
		pixel.Z = n / (vol_ptr->X * vol_ptr->Y) + vol_ptr->Zstart_per + z_start;
		pixel.Y = n % (vol_ptr->X * vol_ptr->Y) / vol_ptr->X + vol_ptr->Ystart;
		pixel.X = n % (vol_ptr->X * vol_ptr->Y) % vol_ptr->X + vol_ptr->Xstart;

		//get the x y of projection
		dev_val_coef(&dev_const_img.pixel0, pixel, ang, zshift,
				dev_const_img.x_coef, dev_const_img.y_coef, &comp_prj);

		//calculate the projection
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& comp_prj.y_min >= 0
				&& comp_prj.y_min < dev_const_img.prj.Y) {
			ASARTatomProjection(comp_prj.x_min, comp_prj.y_min,
					1 - comp_prj.x_min_del, 1 - comp_prj.y_min_del, n);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& comp_prj.y_min >= 0 && comp_prj.y_min < dev_const_img.prj.Y) //(x_min+1, y_min)
						{
			ASARTatomProjection(comp_prj.x_min + 1, comp_prj.y_min,
					comp_prj.x_min_del, 1 - comp_prj.y_min_del, n);
		}
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min, y_min+1)
						{
			ASARTatomProjection(comp_prj.x_min, comp_prj.y_min+1,
					1 - comp_prj.x_min_del, comp_prj.y_min_del, n);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min+1, y_min+1)
						{
			ASARTatomProjection(comp_prj.x_min + 1, comp_prj.y_min + 1,
					comp_prj.x_min_del, comp_prj.y_min_del, n);
		}

	} //end if(n<total_num_slc)
}
__device__ void atomBackProjection(int x, int y, double x_weight,
		double y_weight, int n, float ATOM_ITR_STEP,double &c,double &s) {
	int prj_index;
	prj_index = x + y * dev_const_img.prj.X; //prj index
	double tmp = y_weight * x_weight ;
	double del = dev_const_img.prj_real[prj_index]- dev_const_img.prj_calc[prj_index];
	s+=  tmp * del;
	c+= tmp;
}
__device__ void ASARTatomBackProjection(int x, int y, double x_weight,
		double y_weight, int n, float ATOM_ITR_STEP,double &c,double &s,double &t) {
	int prj_index;
	prj_index = x + y * dev_const_img.prj.X; //prj index
	double tmp = y_weight * x_weight ;
	double del = dev_const_img.prj_real[prj_index]- dev_const_img.prj_calc[prj_index];
	s+=  tmp * del;
	c+= tmp;
	t+=tmp*dev_const_img.prj_calc_abs[prj_index];
}
__global__ void SART_one_update(int ang, float zshift, int z_start,
		int total_num_slc, float ATOM_ITR_STEP) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;
	if (n < total_num_slc) {
		//get the x y z of volume
		Pixel pixel;
		Weight comp_prj;
		Volume *vol_ptr = &(dev_const_img.vol);

		pixel.Z = n / (vol_ptr->X * vol_ptr->Y) + vol_ptr->Zstart_per + z_start;
		pixel.Y = n % (vol_ptr->X * vol_ptr->Y) / vol_ptr->X + vol_ptr->Ystart;
		pixel.X = n % (vol_ptr->X * vol_ptr->Y) % vol_ptr->X + vol_ptr->Xstart;

		//get the x y of projection
		dev_val_coef(&dev_const_img.pixel0, pixel, ang, zshift,
				dev_const_img.x_coef, dev_const_img.y_coef, &comp_prj);

		double c=0;
		double s=0;

		//calculate the projection
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& comp_prj.y_min >= 0
				&& comp_prj.y_min < dev_const_img.prj.Y) {
			atomBackProjection(comp_prj.x_min, comp_prj.y_min,
					1 - comp_prj.x_min_del, 1 - comp_prj.y_min_del, n,
					ATOM_ITR_STEP,c,s);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& comp_prj.y_min >= 0 && comp_prj.y_min < dev_const_img.prj.Y) //(x_min+1, y_min)
						{
			atomBackProjection(comp_prj.x_min + 1, comp_prj.y_min,
					comp_prj.x_min_del, 1 - comp_prj.y_min_del, n,
					ATOM_ITR_STEP,c,s);
		}
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min, y_min+1)
						{
			atomBackProjection(comp_prj.x_min, comp_prj.y_min+1,
					1 - comp_prj.x_min_del, comp_prj.y_min_del, n,
					ATOM_ITR_STEP,c,s);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min+1, y_min+1)
						{
			atomBackProjection(comp_prj.x_min + 1, comp_prj.y_min + 1,
					comp_prj.x_min_del, comp_prj.y_min_del, n, ATOM_ITR_STEP,c,s);
		}
        if(c!=0)
		dev_const_img.slc_data[n]+=(float)(s/c)*ATOM_ITR_STEP;
	} //end if(n<total_num_slc)
}
__global__ void ASART_one_update(int ang, float zshift, int z_start,
		int total_num_slc, float ATOM_ITR_STEP) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;
	if (n < total_num_slc) {
		//get the x y z of volume
		Pixel pixel;
		Weight comp_prj;
		Volume *vol_ptr = &(dev_const_img.vol);

		pixel.Z = n / (vol_ptr->X * vol_ptr->Y) + vol_ptr->Zstart_per + z_start;
		pixel.Y = n % (vol_ptr->X * vol_ptr->Y) / vol_ptr->X + vol_ptr->Ystart;
		pixel.X = n % (vol_ptr->X * vol_ptr->Y) % vol_ptr->X + vol_ptr->Xstart;

		//get the x y of projection
		dev_val_coef(&dev_const_img.pixel0, pixel, ang, zshift,
				dev_const_img.x_coef, dev_const_img.y_coef, &comp_prj);

		double c=0;
		double s=0;
		double t=0;
		//calculate the projection
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& comp_prj.y_min >= 0
				&& comp_prj.y_min < dev_const_img.prj.Y) {
			ASARTatomBackProjection(comp_prj.x_min, comp_prj.y_min,
					1 - comp_prj.x_min_del, 1 - comp_prj.y_min_del, n,
					ATOM_ITR_STEP,c,s,t);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& comp_prj.y_min >= 0 && comp_prj.y_min < dev_const_img.prj.Y) //(x_min+1, y_min)
						{
			ASARTatomBackProjection(comp_prj.x_min + 1, comp_prj.y_min,
					comp_prj.x_min_del, 1 - comp_prj.y_min_del, n,
					ATOM_ITR_STEP,c,s,t);
		}
		if (comp_prj.x_min >= 0 && comp_prj.x_min < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min, y_min+1)
						{
			ASARTatomBackProjection(comp_prj.x_min, comp_prj.y_min+1,
					1 - comp_prj.x_min_del, comp_prj.y_min_del, n,
					ATOM_ITR_STEP,c,s,t);
		}
		if ((comp_prj.x_min + 1) >= 0
				&& (comp_prj.x_min + 1) < dev_const_img.prj.X
				&& (comp_prj.y_min + 1) >= 0
				&& (comp_prj.y_min + 1) < dev_const_img.prj.Y) //(x_min+1, y_min+1)
						{
			ASARTatomBackProjection(comp_prj.x_min + 1, comp_prj.y_min + 1,
					comp_prj.x_min_del, comp_prj.y_min_del, n, ATOM_ITR_STEP,c,s,t);
		}
        if(t!=0)
        	dev_const_img.slc_data[n]+= abs(dev_const_img.slc_data[n]) * (float) s / t * ATOM_ITR_STEP;


	} //end if(n<total_num_slc)
}
__global__ void fixPrjCal(float *prj_calc, float *prj_calc_num, int size_pro) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (prj_calc_num[i] != 0 && i < size_pro) {
		prj_calc[i] = prj_calc[i] / prj_calc_num[i];
	}

//	prj_calc[i] = prj_real[i] - prj_calc[i];
}
void Sart_Slice_reproj(int z_start, int size_slc, int ang, float zshift) {
	int num_blocks = (size_slc - 1) / BLOCKDIM + 1;
	SART_Slice_reproj_kernel<<<num_blocks, BLOCKDIM>>>(ang, zshift, z_start,
			size_slc);
}
void calPrj(float* prj_real, float *prj_calc, float *prj_calc_num,
		int size_pro) {
	int num_blocks = (size_pro - 1) / BLOCKDIM + 1;
	fixPrjCal<<<num_blocks, BLOCKDIM>>>(prj_calc, prj_calc_num, size_pro);
}
void updateSlc(int ang, float zshift, int z_start, int size_slc,
		float ATOM_ITR_STEP) {
	int num_blocks = (size_slc - 1) / BLOCKDIM + 1;
	SART_one_update<<<num_blocks, BLOCKDIM>>>(ang, zshift, z_start, size_slc,
			ATOM_ITR_STEP);
}
void Asart_Slice_reproj(int z_start, int size_slc, int ang, float zshift) {
	int num_blocks = (size_slc - 1) / BLOCKDIM + 1;
	ASART_Slice_reproj_kernel<<<num_blocks, BLOCKDIM>>>(ang, zshift, z_start,
			size_slc);
}

void Asart_updateSlc(int ang, float zshift, int z_start, int size_slc,
		float ATOM_ITR_STEP) {
	int num_blocks = (size_slc - 1) / BLOCKDIM + 1;
	ASART_one_update<<<num_blocks, BLOCKDIM>>>(ang, zshift, z_start, size_slc,
			ATOM_ITR_STEP);
}

