#include "hip/hip_runtime.h"
#include "atomrecon.cuh"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
__constant__ ImageStruct dev_const_img;
__constant__ int slcmin[16];
//#define ICT_NORMAL
long int initGPU(int mpi_rank_id) {

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int gpu_id;
// #ifdef ICT_NORMAL
// 	gpu_id = mpi_rank_id % deviceCount;
// #else
// 	if (mpi_rank_id == 0) {
// 		gpu_id = 0;
// 	} else {
// 		gpu_id = 2;
// 	}
// 
// #endif
#ifdef ICT_NORMAL
	if (mpi_rank_id == 0) {
		gpu_id = 0;
		printf("Using GPU 0\n");
	} else {
		gpu_id = 2;
		printf("Using GPU 2\n");
	}
#else
	gpu_id = mpi_rank_id % deviceCount;
#endif
	hipSetDevice(gpu_id);
	//hipDeviceReset();
	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, gpu_id);

	printf("mpi=%d GPU=%d device properties is :\n"
			"\t device name is %s\n"
			"\t totalGlobalMem is %d MB\n"
			"\t sharedMemPerBlock is %d\n"
			"\t regsPerBlock is %d\n"
			"\t memPitch is %d\n"
			"\t maxThreadsPerBlock is %d\n"
			"\t maxThreadsDim [3] is %d X %d X %d\n"
			"\t maxGridSize [3] is %d X %d X %d\n"
			"\t totalConstMem is %d\n"
			"\t device version is major %d ,minor %d\n"
			"\t clockRate is %d\n"
			"\t textureAlignment is %d\n"
			"\t deviceOverlap is %d\n"
			"\t multiProcessorCount is %d\n", mpi_rank_id, gpu_id,
			device_prop.name, device_prop.totalGlobalMem / 1024 / 1024,
			device_prop.sharedMemPerBlock, device_prop.regsPerBlock,
			device_prop.memPitch, device_prop.maxThreadsPerBlock,
			device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1],
			device_prop.maxThreadsDim[2], device_prop.maxGridSize[0],
			device_prop.maxGridSize[1], device_prop.maxGridSize[2],
			device_prop.totalConstMem, device_prop.major, device_prop.minor,
			device_prop.clockRate, device_prop.textureAlignment,
			device_prop.deviceOverlap, device_prop.multiProcessorCount);
	hipDeviceReset();
	return 0;
}
int initMalloc(ImageStruct & dev_img, int line_num, int prj_AngN,
		int pixel_num,int slicenum) {

	//CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img_ptr, sizeof(ImageStruct)));

	int total_used_mem=0;
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num*slicenum));


	//total_used_mem += sizeof(float) * prj_AngN * 6;

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.cosang,
					sizeof(float) * prj_AngN));
	CUDA_CHECK_RETURN(
				hipMalloc((void ** )&dev_img.sinang,
						sizeof(float) * prj_AngN));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.slc.data, sizeof(float) * pixel_num*slicenum));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.slc.c, sizeof(float) * pixel_num*slicenum));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.slc.s, sizeof(float) * pixel_num*slicenum));
	hipMalloc((void ** )&dev_img.slc.min, sizeof(int) *slicenum);
	hipMalloc((void ** )&dev_img.slc.max, sizeof(int) *slicenum);

//	total_used_mem+=sizeof(float) * line_num *4*slicenum;
//	total_used_mem+=sizeof(float) * prj_AngN ;
	//printf("total_used_mem %d \n",total_used_mem/1024/1024);
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img,sizeof(ImageStruct));
	return 0;
}
void BPT_hostmalloc(float **data,int prjsize,float **slcdata,int slcsize)
{
	hipHostAlloc( (void**)&(*data),prjsize,hipHostMallocDefault);
	hipHostAlloc( (void**)&(*slcdata),slcsize,hipHostMallocDefault);
}
int SIRT_initMalloc(ImageStruct & dev_img,  int line_num,int prj_AngN,int pixel_num,int slicenum
		) {


	//CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img_ptr, sizeof(ImageStruct)));

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num*slicenum));

	//total_used_mem += sizeof(float) * prj_AngN * 6;

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.cosang,sizeof(float) * prj_AngN));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.sinang,sizeof(float) * prj_AngN));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(float)*pixel_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(float)*pixel_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc, sizeof(float)*line_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num, sizeof(float)*line_num*slicenum));
	hipMalloc((void ** )&dev_img.slc.min, sizeof(int) *slicenum);
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}

int SART_initMalloc(ImageStruct & dev_img, int line_num, int prj_AngN,
		int pixel_num,int slicenum) {

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.cosang,sizeof(float) * prj_AngN));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.sinang,sizeof(float) * prj_AngN));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(float)*pixel_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(float)*pixel_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc, sizeof(float)*line_num*slicenum));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num, sizeof(float)*line_num*slicenum));
	hipMalloc((void ** )&dev_img.slc.min, sizeof(int) *slicenum);
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}


void initDevMem(float *dev_data, int size) {

	CUDA_CHECK_RETURN(hipMemset(dev_data, 0, size));
}
void initDevVol(float *dev_data, int size) {

	CUDA_CHECK_RETURN(hipMemset(dev_data, 0, size));
}
void initDevfloat(float *dev_data, int size) {

	CUDA_CHECK_RETURN(hipMemset(dev_data, 0, size));
}
void transferToDevtilt(float *dev_data, float *host_data, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_data, host_data, size, hipMemcpyHostToDevice));
}

void transferToDevPro(float *dev_prj_real, float *prj_real, int size) {

/*	hipEvent_t start1;
		hipEventCreate(&start1);
		hipEvent_t stop1;
		hipEventCreate(&stop1);
		hipEventRecord(start1, NULL);*/
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_prj_real, prj_real, size, hipMemcpyHostToDevice));
/*	 hipEventRecord(stop1, NULL);
		hipEventSynchronize(stop1);
		float msecTotal1 = 0.0f;
		hipEventElapsedTime(&msecTotal1, start1, stop1);
	        printf("copy 1 : %f\n ",msecTotal1);*/
}
void transferIntToDev(int *dev_data, int *host_data, int size) {
	CUDA_CHECK_RETURN(hipMemcpy(dev_data, host_data, size, hipMemcpyHostToDevice));

}
void transferConst(int *host_slcmin,int size)
{

	hipMemcpyToSymbol(HIP_SYMBOL(slcmin), host_slcmin, size);

	//hipMemcpyToSymbol((const void*) &slcmin, (void*) &host_slcmin, size);
}
void transferToDevVol(float *dev_slc_data, float *slc_data, int size) {
/*	hipEvent_t start1;
			hipEventCreate(&start1);
			hipEvent_t stop1;
			hipEventCreate(&stop1);
			hipEventRecord(start1, NULL);*/
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_slc_data, slc_data, size, hipMemcpyHostToDevice));
	/*	 hipEventRecord(stop1, NULL);
			hipEventSynchronize(stop1);
			float msecTotal1 = 0.0f;
			hipEventElapsedTime(&msecTotal1, start1, stop1);
		        printf("copy back : %f\n ",msecTotal1);*/
}
__device__ float lineindex_value(const float &cosang, const float &sinang ,const int &x, const int &z,
		const int &z0, const int &x0) {
	return ((x - x0 + 0.5f) * cosang + (z - z0 + 0.5f) * sinang + x0);
}

__device__ inline void atomicAddF(float* address, float value) {
	while (value)
		value = atomicExch(address, value + atomicExch(address, 0.0f));

}


__global__ void Bpt_Slice_backprj_kernel(int pixel_num, float *prj_real,
		Slice slc, Slide slid,int slicenum,int line_num) {

	const long long n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;

	int realslicenum=n/pixel_num;
/*	if(n==0){
	printf("%d %d %d %d \n",slcmin[0],slcmin[1],slcmin[2],slcmin[3]);
	}*/
	int angindex = blockIdx.y;
	//printf("GPU info --%d  total :%d \n",n,total_num_slc);
	if (n < pixel_num*slicenum) {

		float rf,ang,weight;
		int row_index;
		long long imageindex=n;

	//	pixel.Z = (n) / (dev_const_img.vol.X );
	//	pixel.Y = (n) % (dev_const_img.vol.X * Y_per) / dev_const_img.vol.X;
	//	pixel.X = (n) % dev_const_img.vol.X;

		//imageindex=pixel.X+(j-Y_start)*dev_const_img.vol.X+pixel.Z*dev_const_img.vol.X*Y_per;
		int realindex=n-pixel_num*realslicenum;
		int m = realindex + slcmin[realslicenum];
		//int m = realindex + slc.min[realslicenum];
		int z = m / slid.X;
		int x = m - z * slid.X;
		float temp;

		//ang = D2R(-dev_const_img.tilt_angle[angindex]);
		rf = (x - slid.x0 + 0.5) * dev_const_img.cosang[angindex] + (z - slid.z0 + 0.5) * dev_const_img.sinang[angindex]+ slid.x0;
		int r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//  r+=slid->x0;

		if (r >= 0 && r < slid.X - 1) {
			atomicAdd(&slc.c[imageindex],1.0);
			row_index = angindex * slid.X + r+line_num*realslicenum;
			temp=weight * prj_real[row_index]+ (1 - weight) * prj_real[row_index + 1];
			atomicAdd(&slc.s[imageindex],temp);

		} else if (r == slid.X - 1) {
			atomicAdd(&slc.c[imageindex],weight);
			row_index = angindex * slid.X + r+line_num*realslicenum;
			temp=weight * prj_real[row_index];
			atomicAdd(&slc.s[imageindex],temp);

		} else if (r == -1) {
			atomicAdd(&slc.c[imageindex], 1 - weight);
			row_index = angindex * slid.X+line_num*realslicenum;
			temp= (1 - weight) * prj_real[row_index];
			atomicAdd(&slc.s[imageindex],temp);

		}
	}

}
__global__ void Bpt_Slice_divide_kernel(int pixel_num,Slice slc,int slicenum) {

	long long  n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;

	if (n < pixel_num*slicenum) {

		float tempc=slc.c[n];
		if (tempc != 0){
			slc.data[n] = slc.s[n]/tempc;
			//slc.data[imageindex] = (float)slc.c[imageindex];
		}
			//slc.data[imageindex] = slc.s[imageindex] / slc.c[imageindex];
	}
    /*if (n < pixel_num*slicenum) {
	slc.data[n]=10;
	}*/
}
void transferToHost(float *dev_slc_data,float *slc_data,int size)
{
	CUDA_CHECK_RETURN(
			hipMemcpy(slc_data, dev_slc_data, size ,hipMemcpyDeviceToHost));
}
void Bpt_Slice_backproj(int pixel_num, int prj_angN, long long  Y_per, float *prj_real,
		Slice slc, Slide slid,int slicenum,int line_num) {
/*	hipEvent_t start1;
		hipEventCreate(&start1);
		hipEvent_t stop1;
		hipEventCreate(&stop1);
		hipEventRecord(start1, NULL);*/


	int num_blocks = (pixel_num - 1) / BLOCKDIM + 1;
	dim3 dim2block(num_blocks*slicenum, prj_angN);
	//printf("pixel_num=%d prjnum=%d \n ", pixel_num, prjnum);
	Bpt_Slice_backprj_kernel<<<dim2block, BLOCKDIM>>>(pixel_num, prj_real, slc,
			slid, slicenum,line_num);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());/*
    hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
        printf("kernel 1 : %f\n ",msecTotal1);*/
}
void Bpt_Slice_divide(int total_num_slc,Slice slc,int slicenum) {
	int num_blocks = (total_num_slc - 1) / BLOCKDIM + 1;
/*	hipEvent_t start1;
			hipEventCreate(&start1);
			hipEvent_t stop1;
			hipEventCreate(&stop1);
			hipEventRecord(start1, NULL);
	//printf("pixel_num=%d prjnum=%d \n ", pixel_num, prjnum);*/
	Bpt_Slice_divide_kernel<<<num_blocks*slicenum, BLOCKDIM>>>( total_num_slc,slc,slicenum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

/*	 hipEventRecord(stop1, NULL);
		hipEventSynchronize(stop1);
		float msecTotal1 = 0.0f;
		hipEventElapsedTime(&msecTotal1, start1, stop1);
	        printf("kernel 2 : %f\n ",msecTotal1);*/

}
__global__ void SIRT_Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int AngN,int slicenum,int linenum) //A*X
		{
	int pixel_allnum = dev_const_img.vol.X*dev_const_img.vol.Z;
	const long long  n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	int realslice=n/pixel_allnum;
	long long m;
	int j, z, x, row_index;
	float ang, rf, weight;
	int r;
	//j= blockIdx.y;

//	int pixel_allnum = slc.X * slc.Z;

	if (n < pixel_allnum*slicenum) {
		long long  realindex=n-(realslice*pixel_allnum);
		m = realindex + slcmin[realslice];
		//float *prjreal=prj_real+line_num*realslicenum;

		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		for (j = 0; j < AngN; j++) {
			//ang = PI_180 * c_ang[j];


			rf = lineindex_value(dev_const_img.cosang[j],dev_const_img.sinang[j], x, z, slid.z0, slid.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//r += slc.x0;
			if (r >= 0 && r < slid.X - 1) {
				row_index = j *  slid.X + r+linenum*realslice;
				atomicAdd(&d_prj_num[row_index], weight);
				atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
				atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
				atomicAdd(&d_prj_calc[row_index + 1],(1 - weight) * d_slc_data[n]);
			} //end if weight
			else if (r == slid.X - 1) //left
					{
				row_index = j * slid.X + r+linenum*realslice;
				atomicAdd(&d_prj_num[row_index], weight);
				atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
			} else if (r == -1) //right
					{
				row_index = j * slid.X+linenum*realslice;
				atomicAdd(&d_prj_num[row_index+1], 1 - weight);
				atomicAdd(&d_prj_calc[row_index+1],(1 - weight) * d_slc_data[n]);
			}

		} //end for j
	} //end if m

}

__global__ void proj_calc_div(float *d_prj_calc, float *d_prj_num,
		int line_allnum) {
	const long long  r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	float s, c;
	s = d_prj_calc[r];
	c = d_prj_num[r];
	if ((r < line_allnum) && (c != 0)) {
		d_prj_calc[r] = s/c;
	}
}
__global__ void SIRT_one_update(float *d_prj_real, float *d_prj_calc,
		float *d_slc_data, int AngN, Slice slc, Slide slid, float SIRT_ITR_STEP,
		int slicenum,int linenum) {
	const long long n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	long long m;

	int j, z, x, row_index;
	float c = 0.0f;
	float s = 0.0f;
	float ang, rf, weight;
	int r;

	long long pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	int realslice=n/pixel_allnum;
	//j= blockIdx.y;
	if (n < pixel_allnum*slicenum) {
		long long  realindex=n-(realslice*pixel_allnum);
		m = realindex + slcmin[realslice];


		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		for (j = 0; j < AngN; j++) {

			rf = lineindex_value(dev_const_img.cosang[j],dev_const_img.sinang[j], x, z, slid.z0, slid.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//	r += slc.x0;
			if (r >= 0 && r < slid.X - 1) {
				c++;
				row_index = j * slid.X + r+linenum*realslice;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
				s +=(1 - weight)
								* (d_prj_real[row_index + 1]
										- d_prj_calc[row_index + 1]);
			} //end if
			else if (r == slid.X - 1) {
				c += weight;
				row_index = j * slid.X + r+linenum*realslice;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
			} else if (r == -1) {
				c += 1 - weight;
				row_index = j * slid.X+linenum*realslice;
				s += (1 - weight)
						* (d_prj_real[row_index] - d_prj_calc[row_index]);
			}
		} //end for j

		if (c != 0)
			d_slc_data[n] += __fdividef(s, c) * SIRT_ITR_STEP;
	} //end if m
}
int Cu_SIRT_update_slice(float *d_prj_real, float *d_prj_calc,
		float *d_prj_num, float *d_slc_data, int AngN, Slice slc, Slide slid,
		int pixel_num, int prjX,float SIRT_ITR_STEP,int slicenum) {
	int line_allnum =prjX* AngN *slicenum;
	int num_blocks;
	int linenum=prjX* AngN ;
	//unsigned int timer = 0;
	num_blocks = ((pixel_num *slicenum)+ BLOCKDIM - 1) >> LOG_BLOCKDIM;

	SIRT_Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,d_slc_data, slc, slid, AngN,slicenum,linenum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	num_blocks = (line_allnum + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,
			line_allnum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());


	num_blocks = ((pixel_num *slicenum) + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	SIRT_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_real,d_prj_calc, d_slc_data, AngN, slc, slid, SIRT_ITR_STEP,slicenum,linenum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	return TRUE;
}

__global__ void SART_Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int angleindex,int slicenum,int linenum,int slice) //A*X
		{
	const long long n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	int pixel_allnum = dev_const_img.vol.X*dev_const_img.vol.Z;
	long long m;
	long long z, x, row_index;
	float ang, rf, weight;
	long long r;
	int realslice=n/pixel_allnum;

//	int pixel_allnum = slc.X * slc.Z;

	if (n < pixel_allnum*slicenum) {
		long long  realindex=n-realslice*pixel_allnum;
		m = realindex + slcmin[realslice];
		float *d_prj_num_real=d_prj_num+linenum*realslice;
		float *d_prj_calc_real=d_prj_calc+linenum*realslice;
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;

		rf = lineindex_value(dev_const_img.cosang[angleindex],dev_const_img.sinang[angleindex], x, z, slid.z0, slid.x0);
		r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);

		//r += slc.x0;
		if (r >= 0 && r < slid.X - 1) {
			row_index = angleindex * slid.X + r;

			atomicAdd(&d_prj_num_real[row_index], weight);
			atomicAdd(&d_prj_num_real[row_index + 1], 1 - weight);
			atomicAdd(&d_prj_calc_real[row_index], weight * d_slc_data[n]);
			atomicAdd(&d_prj_calc_real[row_index + 1], (1 - weight) * d_slc_data[n]);

		} //end if weight
		else if (r == slid.X - 1) //left
				{
			row_index = angleindex * slid.X + r;
			atomicAdd(&d_prj_num_real[row_index], weight);
			atomicAdd(&d_prj_calc_real[row_index], weight * d_slc_data[n]);

		} else if (r == -1) //right
				{
			row_index = angleindex * slid.X;
			atomicAdd(&d_prj_num_real[row_index + 1], 1 - weight);
			atomicAdd(&d_prj_calc_real[row_index + 1], (1 - weight) * d_slc_data[n]);

		}

//			printf(" GPU The result: %d  %f\n",n,d_slc_data[n]);
	} //end if n
}
__global__ void SART_proj_calc_div(float *d_prj_calc, float *d_prj_num,
		int angleindex,float *d_prj_real,int slicenum,int prjnum) {
	const long long r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	int realslice=r/dev_const_img.prj.X;
	long long realindex=r-realslice*dev_const_img.prj.X;
	long long prj_index=realindex+angleindex*dev_const_img.prj.X+realslice*prjnum;

	float s, c;
	s = d_prj_calc[prj_index];
	c = d_prj_num[prj_index];
	if ((r < dev_const_img.prj.X*slicenum)) {
		if(c!=0){
			d_prj_calc[prj_index] = __fdividef(s, c);
		}
		d_prj_calc[prj_index]=d_prj_real[prj_index]-d_prj_calc[prj_index];
	}
}
__global__ void SART_one_update(float *d_prj_calc,
		float *d_slc_data, int angleindex, Slice slc, Slide slid, float SART_ITR_STEP,
		int slicenum,int linenum) {
	const long long n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	int pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	long long m;
	long long z, x, row_index;
	float c = 0.0f;
	float s = 0.0f;
	float ang, rf, weight;
	long long r;
	int realslice=n/pixel_allnum;
	int realindex=n-realslice*pixel_allnum;
	m = realindex + slcmin[realslice];

	if (n < pixel_allnum*slicenum) {


		float *d_prj_calc_real=d_prj_calc+linenum*realslice;


		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;

		rf = lineindex_value(dev_const_img.cosang[angleindex],dev_const_img.sinang[angleindex], x, z, slid.z0, slid.x0);
		r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//	r += slc.x0;

		if (r >= 0 && r < slid.X - 1) {
			c++;
			row_index = angleindex * slid.X + r;
			s += weight * (d_prj_calc_real[row_index]);
			s += (1 - weight)
					* (d_prj_calc_real[row_index + 1]);
		} //end if
		else if (r == slid.X - 1) {
			c += weight;
			row_index = angleindex * slid.X + r;
			s += weight * (d_prj_calc_real[row_index]);
		} else if (r == -1) {
			c += 1 - weight;
			row_index = angleindex * slid.X;
			s += (1 - weight) * (d_prj_calc_real[row_index]);
		}

		if (c != 0)
			d_slc_data[n] += __fdividef(s, c) * SART_ITR_STEP;

	} //end if m
}

int Cu_SART_update_slice(float *d_prj_real, float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, int AngN, Slice slc, Slide slid, int pixel_num, int prjX,
		float SART_ITR_STEP ,int slicenum,int slice) {
	int angle;
	int num_blocks;
	int linenum=prjX* AngN ;

	for (angle = 0; angle < AngN; angle++) {

		//unsigned int timer = 0;
		num_blocks = (pixel_num*slicenum + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		SART_Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, d_slc_data, slc, slid, angle, slicenum,linenum,slice);

		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = ((prjX *slicenum)+ BLOCKDIM - 1) >> LOG_BLOCKDIM;
		SART_proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, angle, d_prj_real,slicenum,linenum);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = (pixel_num *slicenum+ BLOCKDIM - 1) >> LOG_BLOCKDIM;
		SART_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(
				d_prj_calc, d_slc_data, angle, slc, slid, SART_ITR_STEP, slicenum,linenum);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

	} // end for angle
	return 0;
}


void freeDevMem_float(float *dev_data) {
	hipFree(dev_data);
}
void freeDevMem_int(int *dev_data) {
	hipFree(dev_data);
}

void freeDevProjection(float *prj) {
	hipFree(prj);
}
void freeDevSlc(float *slc_data) {
	hipFree(slc_data);
}

