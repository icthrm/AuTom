#include "hip/hip_runtime.h"
#include "atomrecon.cuh"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
__constant__ ImageStruct dev_const_img;

long int initGPU(int mpi_rank_id) {

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int gpu_id;
#ifdef ICT_NORMAL
	gpu_id = mpi_rank_id % deviceCount;
#else
	if (mpi_rank_id == 0) {
		gpu_id = 0;
	} else {
		gpu_id = 2;
	}

#endif
	hipSetDevice(gpu_id);
	hipDeviceReset();
	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, gpu_id);

	printf("mpi=%d GPU=%d device properties is :\n"
			"\t device name is %s\n"
			"\t totalGlobalMem is %d MB\n"
			"\t sharedMemPerBlock is %d\n"
			"\t regsPerBlock is %d\n"
			"\t memPitch is %d\n"
			"\t maxThreadsPerBlock is %d\n"
			"\t maxThreadsDim [3] is %d X %d X %d\n"
			"\t maxGridSize [3] is %d X %d X %d\n"
			"\t totalConstMem is %d\n"
			"\t device version is major %d ,minor %d\n"
			"\t clockRate is %d\n"
			"\t textureAlignment is %d\n"
			"\t deviceOverlap is %d\n"
			"\t multiProcessorCount is %d\n", mpi_rank_id, gpu_id,
			device_prop.name, device_prop.totalGlobalMem / 1024 / 1024,
			device_prop.sharedMemPerBlock, device_prop.regsPerBlock,
			device_prop.memPitch, device_prop.maxThreadsPerBlock,
			device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1],
			device_prop.maxThreadsDim[2], device_prop.maxGridSize[0],
			device_prop.maxGridSize[1], device_prop.maxGridSize[2],
			device_prop.totalConstMem, device_prop.major, device_prop.minor,
			device_prop.clockRate, device_prop.textureAlignment,
			device_prop.deviceOverlap, device_prop.multiProcessorCount);
	hipDeviceReset();
	return device_prop.totalGlobalMem;
}
int initMalloc(ImageStruct & dev_img,  int line_num,int prj_AngN,int pixel_num,
		long int total_mem) {
	long int total_used_mem = 0;

	//CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img_ptr, sizeof(ImageStruct)));
	total_used_mem += sizeof(ImageStruct);
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num));

	//total_used_mem += sizeof(double) * prj_AngN * 6;


	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.tilt_angle, sizeof(double)*prj_AngN));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(double)*pixel_num));
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}
int SIRT_initMalloc(ImageStruct & dev_img,  int line_num,int prj_AngN,int pixel_num,
		long int total_mem) {
	long int total_used_mem = 0;

	//CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img_ptr, sizeof(ImageStruct)));
	total_used_mem += sizeof(ImageStruct);
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num));

	//total_used_mem += sizeof(double) * prj_AngN * 6;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.tilt_angle, sizeof(double)*prj_AngN));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc, sizeof(float)*line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num, sizeof(double)*line_num));
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}
int ASIRT_initMalloc(ImageStruct & dev_img,  int line_num,int prj_AngN,int pixel_num,
		long int total_mem) {
	long int total_used_mem = 0;

	total_used_mem += sizeof(ImageStruct);
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.tilt_angle, sizeof(double)*prj_AngN));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc, sizeof(float)*line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num, sizeof(double)*line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num_abs, sizeof(double)*line_num));
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}
int SART_initMalloc(ImageStruct & dev_img, int line_num, int prj_AngN,
		int pixel_num) {

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.tilt_angle, sizeof(double)*prj_AngN));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc, sizeof(float)*line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num, sizeof(double)*line_num));
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}
int ASART_initMalloc(ImageStruct & dev_img, int line_num, int prj_AngN,
		int pixel_num) {

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_img.prj_real, sizeof(float) * line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.tilt_angle, sizeof(double)*prj_AngN));

	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.data, sizeof(float)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.c, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.slc.s, sizeof(double)*pixel_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc, sizeof(float)*line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num, sizeof(double)*line_num));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&dev_img.prj_calc_num_abs, sizeof(double)*line_num));
	hipMemcpyToSymbol((const void*) &dev_const_img, (void*) &dev_img, sizeof(ImageStruct));
	return 0;
}

void initDevMem(double *dev_data, int size) {

	CUDA_CHECK_RETURN(hipMemset(dev_data, 0, size));
}
void initDevVol(float *dev_data, int size) {

	CUDA_CHECK_RETURN(hipMemset(dev_data, 0, size));
}
void initDevfloat(float *dev_data, int size) {

	CUDA_CHECK_RETURN(hipMemset(dev_data, 0, size));
}
void transferToDevtilt(double *dev_data, double *host_data, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_data, host_data, size, hipMemcpyHostToDevice));
}

void transferToDevPro(float *dev_prj_real, float *prj_real, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_prj_real, prj_real, size, hipMemcpyHostToDevice));
}
void transferToDevVol(float *dev_slc_data, float *slc_data, int size) {
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_slc_data, slc_data, size, hipMemcpyHostToDevice));
}
__device__ double lineindex_value(const double &ang, const int &x, const int &z,
		const int &z0, const int &x0) {
	return ((x - x0 + 0.5f) * __cosf(ang) + (z - z0 + 0.5f) * __sinf(ang) + x0);
}
__device__ inline void atomicAddF(float* address, float value) {
	while (value)
		value = atomicExch(address, value + atomicExch(address, 0.0f));

}
__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__global__ void Bpt_Slice_backprj_kernel(int total_num_slc, int Y_per, float *prj_real,
		Slice slc, Slide slid,int Y_start,int j) {

	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;
	int angindex = blockIdx.y;
	//printf("GPU info --%d  total :%d \n",n,total_num_slc);
	if (n < total_num_slc) {
		double rf,ang,weight;
		int row_index;
		int imageindex;

	//	pixel.Z = (n) / (dev_const_img.vol.X );
	//	pixel.Y = (n) % (dev_const_img.vol.X * Y_per) / dev_const_img.vol.X;
	//	pixel.X = (n) % dev_const_img.vol.X;

		//imageindex=pixel.X+(j-Y_start)*dev_const_img.vol.X+pixel.Z*dev_const_img.vol.X*Y_per;
		imageindex=n;
		int m = n + slc.min;
		int z = m / slid.X;
		int x = m - z * slid.X;
		double temp;
		ang = D2R(-dev_const_img.tilt_angle[angindex]);

		rf = (x - slid.x0 + 0.5) * cos(ang) + (z - slid.z0 + 0.5) * sin(ang)
				+ slid.x0;
		int r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//  r+=slid->x0;
		if(j==100&&n==0&&(angindex==6||angindex==7))
			printf("Test -weight-GPU-weight %f r %d %f %f\n",weight,r,rf,ang);
		if (r >= 0 && r < slid.X - 1) {
			atomicAdd(&slc.c[imageindex],1.0);
			row_index = angindex * slid.X + r;
			temp=weight * prj_real[row_index]+ (1 - weight) * prj_real[row_index + 1];
			atomicAdd(&slc.s[imageindex],temp);
			if(j==100&&n==0)
				 printf("Test -0-0-1-GPU- %d %f %f %f %f\n",row_index,prj_real[row_index],prj_real[row_index+1],slc.s[n],weight);
		} else if (r == slid.X - 1) {
			atomicAdd(&slc.c[imageindex],weight);
			row_index = angindex * slid.X + r;
			temp=weight * prj_real[row_index];
			atomicAdd(&slc.s[imageindex],temp);
			if(j==100&&n==0)
						 printf("Test -0-0-2-GPU- %d %f %f %f\n",row_index,prj_real[row_index],prj_real[row_index+1],slc.s[n]);
		} else if (r == -1) {
			atomicAdd(&slc.c[imageindex], 1 - weight);
			row_index = angindex * slid.X;
			temp= (1 - weight) * prj_real[row_index];
			atomicAdd(&slc.s[imageindex],temp);
			if(j==100&&n==0)
						 printf("Test -0-0-3-GPU- %d %f %f %f\n",row_index,prj_real[row_index],prj_real[row_index+1],slc.s[n]);
		}
	}

}
__global__ void Bpt_Slice_divide_kernel(int total_num_slc,Slice slc,int Y_start,int j,int Y_per) {

	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x;

	if (n < total_num_slc) {

		Pixel pixel;
		int imageindex;
		pixel.Z = (n) / (dev_const_img.vol.X);
		//	pixel.Y = (n) % (dev_const_img.vol.X * Y_per) / dev_const_img.vol.X;
		pixel.X = (n) % dev_const_img.vol.X;

		imageindex = pixel.X + (j - Y_start) * dev_const_img.vol.X
				+ pixel.Z * dev_const_img.vol.X * Y_per;
		imageindex=n;
		if (slc.c[imageindex] != 0){
			slc.data[imageindex] = slc.s[imageindex]/slc.c[imageindex];
			//slc.data[imageindex] = (float)slc.c[imageindex];
		}
			//slc.data[imageindex] = slc.s[imageindex] / slc.c[imageindex];
	}
}
void transferToHost(float *dev_slc_data,float *slc_data,int size)
{
	CUDA_CHECK_RETURN(
			hipMemcpy(slc_data, dev_slc_data, size ,hipMemcpyDeviceToHost));
}
void Bpt_Slice_backproj(int pixel_num, int prj_angN, int Y_per, float *prj_real,
		Slice slc, Slide slid,int Y_start,int j) {
	int num_blocks = (pixel_num - 1) / BLOCKDIM + 1;
	dim3 dim2block(num_blocks, prj_angN);
	//printf("pixel_num=%d prjnum=%d \n ", pixel_num, prjnum);
	Bpt_Slice_backprj_kernel<<<dim2block, BLOCKDIM>>>(pixel_num, Y_per, prj_real, slc,
			slid,Y_start,j);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
}
void Bpt_Slice_divide(int total_num_slc,Slice slc,int Y_start,int j,int Y_per) {
	int num_blocks = (total_num_slc - 1) / BLOCKDIM + 1;
	//printf("pixel_num=%d prjnum=%d \n ", pixel_num, prjnum);
	Bpt_Slice_divide_kernel<<<num_blocks, BLOCKDIM>>>( total_num_slc,slc,Y_start,j, Y_per);

}
__global__ void SIRT_Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int AngN,int jslide) //A*X
		{
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	double ang, rf, weight;
	int r;
	m = n + slc.min;
//	int pixel_allnum = slc.X * slc.Z;
	int pixel_allnum = dev_const_img.vol.X*dev_const_img.vol.Z;
	if (n < pixel_allnum) {

		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		for (j = 0; j < AngN; j++) {
			//ang = PI_180 * c_ang[j];
	        ang = D2R(-dev_const_img.tilt_angle[j]);
			rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//r += slc.x0;
			if (r >= 0 && r < slid.X - 1) {
				row_index = j *  slid.X + r;
				atomicAdd(&d_prj_num[row_index], weight);
				atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
				atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
				atomicAdd(&d_prj_calc[row_index + 1],
						(1 - weight) * d_slc_data[n]);
			} //end if weight
			else if (r == slid.X - 1) //left
					{
				row_index = j * slid.X + r;
				atomicAdd(&d_prj_num[row_index], weight);
				atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
			} else if (r == -1) //right
					{
				row_index = j * slid.X;
				atomicAdd(&d_prj_num[row_index+1], 1 - weight);
				atomicAdd(&d_prj_calc[row_index+1],
						(1 - weight) * d_slc_data[n]);
			}

		} //end for j
	} //end if m

}
__global__ void proj_calc_div(float *d_prj_calc, float *d_prj_num,
		int line_allnum) {
	const unsigned int r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	double s, c;
	s = d_prj_calc[r];
	c = d_prj_num[r];
	if ((r < line_allnum) && (c != 0)) {
		d_prj_calc[r] = s/c;
	}
}
__global__ void SIRT_one_update(float *d_prj_real, float *d_prj_calc,
		float *d_slc_data, int AngN, Slice slc, Slide slid, float SIRT_ITR_STEP,
		int jslide) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	double c = 0.0f;
	double s = 0.0f;
	double ang, rf, weight;
	int r;
	m = n + slc.min;
	int pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	if (n < pixel_allnum) {
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		for (j = 0; j < AngN; j++) {
			ang = D2R(-dev_const_img.tilt_angle[j]);
			rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//	r += slc.x0;
			if (r >= 0 && r < slid.X - 1) {
				c++;
				row_index = j * slid.X + r;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
				s +=
						(1 - weight)
								* (d_prj_real[row_index + 1]
										- d_prj_calc[row_index + 1]);
			} //end if
			else if (r == slid.X - 1) {
				c += weight;
				row_index = j * slid.X + r;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
			} else if (r == -1) {
				c += 1 - weight;
				row_index = j * slid.X;
				s += (1 - weight)
						* (d_prj_real[row_index] - d_prj_calc[row_index]);
			}
		} //end for j

		if (c != 0)
			d_slc_data[n] += __fdividef(s, c) * SIRT_ITR_STEP;
	} //end if m
}
int Cu_SIRT_update_slice(float *d_prj_real, float *d_prj_calc,
		float *d_prj_num, float *d_slc_data, int AngN, Slice slc, Slide slid,
		int pixel_num, float SIRT_ITR_STEP,ImageStruct host_img,int jslide) {
	int line_allnum =host_img.prj.X* AngN ;
	int num_blocks;
	//unsigned int timer = 0;
	num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	SIRT_Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,
			d_slc_data, slc, slid, AngN,jslide);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	num_blocks = (line_allnum + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,
			line_allnum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	SIRT_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_real,
			d_prj_calc, d_slc_data, AngN, slc, slid, SIRT_ITR_STEP,jslide);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	return TRUE;
}
__global__ void ASIRT_Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int AngN, int jslide,
		float *d_prj_calc_abs) //A*X
		{
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	float ang, rf, weight;
	int r;
	m = n + slc.min;
//	int pixel_allnum = slc.X * slc.Z;
	int pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	if (n < pixel_allnum) {

		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		for (j = 0; j < AngN; j++) {
			//ang = PI_180 * c_ang[j];
			ang = D2R(-dev_const_img.tilt_angle[j]);
			rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//r += slc.x0;
			if (r >= 0 && r < slid.X - 1) {
				row_index = j * slid.X + r;
				atomicAdd(&d_prj_num[row_index], weight);
				atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
				atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
				atomicAdd(&d_prj_calc[row_index + 1],
						(1 - weight) * d_slc_data[n]);
				atomicAdd(&d_prj_calc_abs[row_index],
						weight * abs(d_slc_data[n]));
				atomicAdd(&d_prj_calc_abs[row_index + 1],
						(1 - weight) * abs(d_slc_data[n]));

			} //end if weight
			else if (r == slid.X - 1) //left
					{
				row_index = j * slid.X + r;
				atomicAdd(&d_prj_num[row_index], weight);
				atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
				atomicAdd(&d_prj_calc_abs[row_index],
						weight * abs(d_slc_data[n]));

			} else if (r == -1) //right
					{
				row_index = j * slid.X;
				atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
				atomicAdd(&d_prj_calc[row_index + 1],
						(1 - weight) * d_slc_data[n]);
				atomicAdd(&d_prj_calc_abs[row_index],
						(1 - weight) * abs(d_slc_data[n]));
			}
		} //end for j
	} //end if m
}
__global__ void ASIRT_proj_calc_div(float *d_prj_calc, float *d_prj_num,float *d_prj_calc_abs,
		int line_allnum) {
	const unsigned int r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	float s, c,s2;
	s = d_prj_calc[r];
	c = d_prj_num[r];
	s2=d_prj_calc_abs[r];
	if ((r < line_allnum) && (c != 0)) {
		d_prj_calc[r] = __fdividef(s, c);
		d_prj_calc_abs[r] = __fdividef(s2, c);
	}
}
__global__ void ASIRT_one_update(float *d_prj_real, float *d_prj_calc,
		float *d_slc_data, int AngN, Slice slc, Slide slid, float ASIRT_ITR_STEP,
		int jslide,float *d_prj_calc_abs) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int j, z, x, row_index;
	float c = 0.0f;
	float s = 0.0f;
	float t=0;
	float ang, rf, weight;
	int r;
	m = n + slc.min;
	int pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	if (n < pixel_allnum) {
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		for (j = 0; j < AngN; j++) {
			ang = D2R(-dev_const_img.tilt_angle[j]);
			rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
			r = floor(rf - 0.5);
			weight = 1.5 - (rf - r);
			//	r += slc.x0;
			if (r >= 0 && r < slid.X - 1) {
				c++;
				row_index = j * slid.X + r;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
				s +=(1 - weight)* (d_prj_real[row_index + 1]- d_prj_calc[row_index + 1]);
				t +=weight*d_prj_calc_abs[row_index]+(1-weight)*d_prj_calc_abs[row_index+1];
			} //end if
			else if (r == slid.X - 1) {
				c += weight;
				row_index = j * slid.X + r;
				s += weight * (d_prj_real[row_index] - d_prj_calc[row_index]);
				t+=weight*d_prj_calc_abs[row_index];
			} else if (r == -1) {
				c += 1 - weight;
				row_index = j * slid.X;
				s += (1 - weight)
						* (d_prj_real[row_index] - d_prj_calc[row_index]);
		         t+=(1-weight)*d_prj_calc_abs[row_index];
			}
		} //end for j

		if (t != 0)
			d_slc_data[n] += abs(d_slc_data[n])*(float)s/t*ASIRT_ITR_STEP;
	} //end if m
}
int Cu_ASIRT_update_slice(float *d_prj_real, float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, int AngN, Slice slc, Slide slid, int pixel_num,
		float ASIRT_ITR_STEP, ImageStruct host_img, int jslide,float *d_prj_calc_abs) {
	int line_allnum =host_img.prj.X* AngN ;
	int num_blocks;
	//unsigned int timer = 0;
	num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	ASIRT_Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,
			d_slc_data, slc, slid, AngN,jslide,d_prj_calc_abs);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	num_blocks = (line_allnum + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	ASIRT_proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc, d_prj_num,d_prj_calc_abs,
			line_allnum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
	ASIRT_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_real,d_prj_calc, d_slc_data, AngN, slc, slid, ASIRT_ITR_STEP,jslide,d_prj_calc_abs);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	return TRUE;
}

__global__ void SART_Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int angleindex,int jslide) //A*X
		{
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int z, x, row_index;
	double ang, rf, weight;
	int r;
	m = n + slc.min;
//	int pixel_allnum = slc.X * slc.Z;
	int pixel_allnum = dev_const_img.vol.X*dev_const_img.vol.Z;
	if (n < pixel_allnum) {
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		ang = D2R(-dev_const_img.tilt_angle[angleindex]);
		rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
		r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//r += slc.x0;
		if (r >= 0 && r < slid.X - 1) {
			row_index = angleindex * slid.X + r;
			atomicAdd(&d_prj_num[row_index], weight);
			atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
			atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
			atomicAdd(&d_prj_calc[row_index + 1], (1 - weight) * d_slc_data[n]);
		} //end if weight
		else if (r == slid.X - 1) //left
				{
			row_index = angleindex * slid.X + r;
			atomicAdd(&d_prj_num[row_index], weight);
			atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
		} else if (r == -1) //right
				{
			row_index = angleindex * slid.X;
			atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
			atomicAdd(&d_prj_calc[row_index + 1], (1 - weight) * d_slc_data[n]);
		}
//		if(row_index==800&&jslide==3)
//			printf(" GPU The result: %d  %f\n",n,d_slc_data[n]);
	} //end if n
}
__global__ void SART_proj_calc_div(float *d_prj_calc, float *d_prj_num,
		int angleindex,float *d_prj_real,int jslide) {
	const unsigned int r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	int prj_index=r+angleindex*dev_const_img.prj.X;
	double s, c;
	s = d_prj_calc[prj_index];
	c = d_prj_num[prj_index];
	if ((r < dev_const_img.prj.X)) {
		if(c!=0){
			d_prj_calc[prj_index] = __fdividef(s, c);
		}
		d_prj_calc[prj_index]=d_prj_real[prj_index]-d_prj_calc[prj_index];
	}
}
__global__ void SART_one_update(float *d_prj_calc,
		float *d_slc_data, int angleindex, Slice slc, Slide slid, float SIRT_ITR_STEP,
		int jslide) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int z, x, row_index;
	double c = 0.0f;
	double s = 0.0f;
	double ang, rf, weight;
	int r;
	m = n + slc.min;
	int pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	if (n < pixel_allnum) {
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		ang = D2R(-dev_const_img.tilt_angle[angleindex]);
		rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
		r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//	r += slc.x0;

		if (r >= 0 && r < slid.X - 1) {
			c++;
			row_index = angleindex * slid.X + r;
			s += weight * (d_prj_calc[row_index]);
			s += (1 - weight)
					* (d_prj_calc[row_index + 1]);
		} //end if
		else if (r == slid.X - 1) {
			c += weight;
			row_index = angleindex * slid.X + r;
			s += weight * (d_prj_calc[row_index]);
		} else if (r == -1) {
			c += 1 - weight;
			row_index = angleindex * slid.X;
			s += (1 - weight) * (d_prj_calc[row_index]);
		}

		if (c != 0)
			d_slc_data[n] += __fdividef(s, c) * SIRT_ITR_STEP;
	} //end if m
}
int Cu_SART_update_slice(float *d_prj_real, float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, int AngN, Slice slc, Slide slid, int pixel_num,
		float SIRT_ITR_STEP, ImageStruct host_img, int jslide) {
	int angle;
	for (angle = 0; angle < host_img.prj.AngN; angle++) {
		int num_blocks;
		//unsigned int timer = 0;
		num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		SART_Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, d_slc_data, slc, slid, angle, jslide);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = (host_img.prj.X + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		SART_proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, angle, d_prj_real,jslide);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		SART_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(
				d_prj_calc, d_slc_data, angle, slc, slid, SIRT_ITR_STEP, jslide);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	} // end for angle
	return 0;
}
__global__ void ASART_Slice_reproj(float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, Slice slc, Slide slid, int angleindex,int jslide,float *d_prj_calc_abs) //A*X
		{
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int z, x, row_index;
	double ang, rf, weight;
	int r;
	m = n + slc.min;
//	int pixel_allnum = slc.X * slc.Z;
	int pixel_allnum = dev_const_img.vol.X*dev_const_img.vol.Z;
	if (n < pixel_allnum) {
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		ang = D2R(-dev_const_img.tilt_angle[angleindex]);
		rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
		r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//r += slc.x0;
		if (r >= 0 && r < slid.X - 1) {
			row_index = angleindex * slid.X + r;
			atomicAdd(&d_prj_num[row_index], weight);
			atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
			atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
			atomicAdd(&d_prj_calc[row_index + 1], (1 - weight) * d_slc_data[n]);
			atomicAdd(&d_prj_calc_abs[row_index],
					weight * abs(d_slc_data[n]));
			atomicAdd(&d_prj_calc_abs[row_index + 1],
					(1 - weight) * abs(d_slc_data[n]));
		} //end if weight
		else if (r == slid.X - 1) //left
				{
			row_index = angleindex * slid.X + r;
			atomicAdd(&d_prj_num[row_index], weight);
			atomicAdd(&d_prj_calc[row_index], weight * d_slc_data[n]);
			atomicAdd(&d_prj_calc_abs[row_index],
							weight * abs(d_slc_data[n]));
		} else if (r == -1) //right
				{
			row_index = angleindex * slid.X;
			atomicAdd(&d_prj_num[row_index + 1], 1 - weight);
			atomicAdd(&d_prj_calc[row_index + 1], (1 - weight) * d_slc_data[n]);
			atomicAdd(&d_prj_calc_abs[row_index],
								(1-weight) * abs(d_slc_data[n]));
		}
	} //end if n
}
__global__ void ASART_proj_calc_div(float *d_prj_calc, float *d_prj_num,
		int angleindex,float *d_prj_real,int jslide,float *d_prj_calc_abs) {
	const unsigned int r = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //prj index
	int prj_index=r+angleindex*dev_const_img.prj.X;
	double s, c,s2;
	s = d_prj_calc[prj_index];
	c = d_prj_num[prj_index];
	s2=d_prj_calc_abs[prj_index];
	if ((r < dev_const_img.prj.X)) {
		if(c!=0){
			d_prj_calc[prj_index] = __fdividef(s, c);
			d_prj_calc_abs[prj_index]= __fdividef(s2, c);
		}
		d_prj_calc[prj_index]=d_prj_real[prj_index]-d_prj_calc[prj_index];
	}
}
__global__ void ASART_one_update(float *d_prj_calc,
		float *d_slc_data, int angleindex, Slice slc, Slide slid, float ASART_ITR_STEP,
		int jslide,float *d_prj_calc_abs) {
	const unsigned int n = (blockIdx.x << LOG_BLOCKDIM) + threadIdx.x; //pixel index and thread index
	unsigned int m;
	int z, x, row_index;
	double c = 0.0f;
	double s = 0.0f;
	double t=0;
	double ang, rf, weight;
	int r;
	m = n + slc.min;
	int pixel_allnum = dev_const_img.vol.X * dev_const_img.vol.Z;
	if (n < pixel_allnum) {
		z = (int) __fdividef(m, slid.X);
		x = m - z * slid.X;
		ang = D2R(-dev_const_img.tilt_angle[angleindex]);
		rf = lineindex_value(ang, x, z, slid.z0, slid.x0);
		r = floor(rf - 0.5);
		weight = 1.5 - (rf - r);
		//	r += slc.x0;

		if (r >= 0 && r < slid.X - 1) {
			c++;
			row_index = angleindex * slid.X + r;
			s += weight * (d_prj_calc[row_index]);
			s += (1 - weight)
					* (d_prj_calc[row_index + 1]);
			t+=weight*d_prj_calc_abs[row_index]+(1-weight)*d_prj_calc_abs[row_index+1];
		} //end if
		else if (r == slid.X - 1) {
			c += weight;
			row_index = angleindex * slid.X + r;
			s += weight * (d_prj_calc[row_index]);
		     t+=weight*d_prj_calc_abs[row_index];
		} else if (r == -1) {
			c += 1 - weight;
			row_index = angleindex * slid.X;
			s += (1 - weight) * (d_prj_calc[row_index]);
		     t+=(1-weight)*d_prj_calc_abs[row_index];
		}
		if(t!=0)    d_slc_data[n] += abs(d_slc_data[n])*(float)s/t*ASART_ITR_STEP;
	} //end if m
}
int Cu_ASART_update_slice(float *d_prj_real, float *d_prj_calc, float *d_prj_num,
		float *d_slc_data, int AngN, Slice slc, Slide slid, int pixel_num,
		float ASART_ITR_STEP, ImageStruct host_img, int jslide,float *d_prj_calc_abs) {
	int angle;
	for (angle = 0; angle < host_img.prj.AngN; angle++) {
		int num_blocks;
		//unsigned int timer = 0;
		num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		ASART_Slice_reproj<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, d_slc_data, slc, slid, angle, jslide,d_prj_calc_abs);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = (host_img.prj.X + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		ASART_proj_calc_div<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(d_prj_calc,
				d_prj_num, angle, d_prj_real,jslide,d_prj_calc_abs);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		num_blocks = (pixel_num + BLOCKDIM - 1) >> LOG_BLOCKDIM;
		ASART_one_update<<<dim3(num_blocks), dim3(BLOCKDIM)>>>(
				d_prj_calc, d_slc_data, angle, slc, slid, ASART_ITR_STEP, jslide,d_prj_calc_abs);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	} // end for angle
	return 0;
}

void freeDevMem_double(double *dev_data) {
	hipFree(dev_data);
}
void freeDevMem_float(double *dev_data) {

	hipFree(dev_data);
}
void freeDevProjection(float *prj) {
	hipFree(prj);
}
void freeDevSlc(float *slc_data) {
	hipFree(slc_data);
}

